#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <string>
#include <vector>
#include <algorithm>

#include <mirheo/core/logger.h>
#include <mirheo/core/task_scheduler.h>

#include "../timer.h"

#define private public

using namespace mirheo;

void verifyDep(const std::string& before, const std::string& after,
               const std::vector<std::string>& messages)
{
    auto itb = std::find(messages.begin(), messages.end(), before);
    auto ita = std::find(messages.begin(), messages.end(), after);

    ASSERT_NE(itb, messages.end());
    ASSERT_NE(ita, messages.end());
    ASSERT_LT(itb, ita);
}

TEST(Scheduler, Order)
{
    /*
      A1,A2 - B -----------
                  \        \
                    D1,D2 - E
              C - /
                  \ F
                            G
    */

    TaskScheduler scheduler;
    std::vector<std::string> messages;

    auto A1 = scheduler.createTask("A1");
    auto A2 = scheduler.createTask("A2");
    auto B  = scheduler.createTask("B");
    auto C  = scheduler.createTask("C");
    auto D1 = scheduler.createTask("D1");
    auto D2 = scheduler.createTask("D2");
    auto E  = scheduler.createTask("E");
    auto F  = scheduler.createTask("F");
    auto G  = scheduler.createTask("G");

    scheduler.addTask(A1, [&](__UNUSED hipStream_t s){ messages.push_back("a1"); });
    scheduler.addTask(A2, [&](__UNUSED hipStream_t s){ messages.push_back("a2"); });
    scheduler.addTask(B , [&](__UNUSED hipStream_t s){ messages.push_back("b" ); });
    scheduler.addTask(C , [&](__UNUSED hipStream_t s){ messages.push_back("c" ); });
    scheduler.addTask(D1, [&](__UNUSED hipStream_t s){ messages.push_back("d1"); });
    scheduler.addTask(D2, [&](__UNUSED hipStream_t s){ messages.push_back("d2"); });
    scheduler.addTask(E , [&](__UNUSED hipStream_t s){ messages.push_back("e" ); });
    scheduler.addTask(F , [&](__UNUSED hipStream_t s){ messages.push_back("f" ); });
    scheduler.addTask(G , [&](__UNUSED hipStream_t s){ messages.push_back("g" ); });

    scheduler.addDependency(B, {}, {A1, A2});
    scheduler.addDependency(D1, {}, {B, C});
    scheduler.addDependency(D2, {}, {B, C});
    scheduler.addDependency(F, {}, {C});
    scheduler.addDependency(E, {}, {D1, D2, B});

    scheduler.compile();
    scheduler.run();

    ASSERT_EQ(messages.size(), 9);

    verifyDep("a1", "b", messages);
    verifyDep("a2", "b", messages);

    verifyDep("b", "d1", messages);
    verifyDep("c", "d1", messages);

    verifyDep("b", "d2", messages);
    verifyDep("c", "d2", messages);

    verifyDep("c", "f", messages);

    verifyDep("d1", "e", messages);
    verifyDep("d2", "e", messages);
    verifyDep("b" , "e", messages);
}

TEST(Scheduler, Benchmark)
{
    TaskScheduler scheduler;

    float a, b, c, d, e, f, g;
    a = b = c = d = e = f = g = 0;

    auto A1 = scheduler.createTask("A1");
    auto A2 = scheduler.createTask("A2");
    auto B  = scheduler.createTask("B");
    auto C  = scheduler.createTask("C");
    auto D1 = scheduler.createTask("D1");
    auto D2 = scheduler.createTask("D2");
    auto E  = scheduler.createTask("E");
    auto F  = scheduler.createTask("F");
    auto G  = scheduler.createTask("G");

    scheduler.addTask(C,  [&](__UNUSED hipStream_t s){ c++; });
    scheduler.addTask(G,  [&](__UNUSED hipStream_t s){ g--; });
    scheduler.addTask(D1, [&](__UNUSED hipStream_t s){ d+=2; });
    scheduler.addTask(A1, [&](__UNUSED hipStream_t s){ a-=3; });
    scheduler.addTask(E,  [&](__UNUSED hipStream_t s){ e*=1.001; });
    scheduler.addTask(A2, [&](__UNUSED hipStream_t s){ a*=0.9999; });
    scheduler.addTask(B,  [&](__UNUSED hipStream_t s){ b+=5; });
    scheduler.addTask(D2, [&](__UNUSED hipStream_t s){ d-=42; });
    scheduler.addTask(F,  [&](__UNUSED hipStream_t s){ f*=2; });

    scheduler.addDependency(B, {}, {A1, A2});
    scheduler.addDependency(D1, {}, {B, C});
    scheduler.addDependency(D2, {}, {B, C});
    scheduler.addDependency(F, {}, {C});
    scheduler.addDependency(E, {}, {D1, D2, B});

    scheduler.compile();

    Timer timer;
    timer.start();

    int n = 10000;
    for (int i=0; i<n; i++)
        scheduler.run();

    int64_t tm = timer.elapsed();

    double tus = (double)tm / (1000.0*n);
    fprintf(stderr, "Per run: %f us\n", tus);

    EXPECT_LE(tus, 500.0);
}

int main(int argc, char **argv)
{
    int provided;
    MPI_Init_thread(&argc, &argv, MPI_THREAD_MULTIPLE, &provided);
    if (provided < MPI_THREAD_MULTIPLE) {
        fprintf(stderr, "ERROR: The MPI library does not have full thread support\n");
        MPI_Abort(MPI_COMM_WORLD, 1);
    }
    MPI_Errhandler_set(MPI_COMM_WORLD, MPI_ERRORS_RETURN);
    logger.init(MPI_COMM_WORLD, "scheduler.log", 9);

    testing::InitGoogleTest(&argc, argv);

    auto ret = RUN_ALL_TESTS();

    MPI_Finalize();
    return ret;
}
