#include "hip/hip_runtime.h"
// Copyright 2020 ETH Zurich. All Rights Reserved.
#include <mirheo/core/utils/restart_helpers.h>

#include "velocity_control.h"
#include "utils/simple_serializer.h"
#include "utils/time_stamp.h"

#include <mirheo/core/datatypes.h>
#include <mirheo/core/pvs/particle_vector.h>
#include <mirheo/core/pvs/views/pv.h>
#include <mirheo/core/simulation.h>
#include <mirheo/core/utils/cuda_common.h>
#include <mirheo/core/utils/kernel_launch.h>
#include <mirheo/core/utils/path.h>

namespace mirheo
{

namespace velocity_control_kernels
{

inline __device__ bool is_inside(real3 r, real3 low, real3 high)
{
    return
        low.x <= r.x && r.x <= high.x &&
        low.y <= r.y && r.y <= high.y &&
        low.z <= r.z && r.z <= high.z;
}

__global__ void addForce(PVview view, DomainInfo domain, real3 low, real3 high, real3 force)
{
    const int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid >= view.size) return;

    auto r = Real3_int(view.readPosition(gid)).v;

    const real3 gr = domain.local2global(r);

    if (is_inside(gr, low, high))
        view.forces[gid] += make_real4(force, 0.0_r);
}

__global__ void sumVelocity(PVview view, DomainInfo domain, real3 low, real3 high, real3 *totVel, int *nSamples)
{
    const int gid = blockIdx.x * blockDim.x + threadIdx.x;
    Particle p;

    p.u = make_real3(0.0_r);

    if (gid < view.size) {

        p = view.readParticle(gid);
        const real3 gr = domain.local2global(p.r);

        if (is_inside(gr, low, high))
            atomicAggInc(nSamples);
        else
            p.u = make_real3(0.0_r);
    }

    const real3 u = warpReduce(p.u, [](real a, real b) { return a+b; });

    if (laneId() == 0 && dot(u, u) > 1e-8)
        atomicAdd(totVel, u);
}

} // namespace velocity_control_kernels

SimulationVelocityControl::SimulationVelocityControl(const MirState *state, std::string name, std::vector<std::string> pvNames,
                                                     real3 low, real3 high,
                                                     int sampleEvery, int tuneEvery, int dumpEvery,
                                                     real3 targetVel, real Kp, real Ki, real Kd) :
    SimulationPlugin(state, name),
    pvNames_(pvNames),
    low_(low),
    high_(high),
    currentVel_(make_real3(0,0,0)),
    targetVel_(targetVel),
    sampleEvery_(sampleEvery),
    tuneEvery_(tuneEvery),
    dumpEvery_(dumpEvery),
    force_(make_real3(0, 0, 0)),
    pid_(make_real3(0, 0, 0), Kp, Ki, Kd),
    accumulatedTotVel_({0,0,0})
{}


void SimulationVelocityControl::setup(Simulation* simulation, const MPI_Comm& comm, const MPI_Comm& interComm)
{
    SimulationPlugin::setup(simulation, comm, interComm);

    pvs_.clear();
    for (auto &pvName : pvNames_)
        pvs_.push_back(simulation->getPVbyNameOrDie(pvName));
}

void SimulationVelocityControl::beforeForces(hipStream_t stream)
{
    for (auto &pv : pvs_)
    {
        PVview view(pv, pv->local());
        const int nthreads = 128;

        SAFE_KERNEL_LAUNCH
            (velocity_control_kernels::addForce,
             getNblocks(view.size, nthreads), nthreads, 0, stream,
             view, getState()->domain, low_, high_, force_ );
    }
}

void SimulationVelocityControl::_sampleOnePv(ParticleVector *pv, hipStream_t stream) {
    PVview pvView(pv, pv->local());
    const int nthreads = 128;

    SAFE_KERNEL_LAUNCH
        (velocity_control_kernels::sumVelocity,
         getNblocks(pvView.size, nthreads), nthreads, 0, stream,
         pvView, getState()->domain, low_, high_, totVel_.devPtr(), nSamples_.devPtr());
}

void SimulationVelocityControl::afterIntegration(hipStream_t stream)
{
    if (isTimeEvery(getState(), sampleEvery_))
    {
        debug2("Velocity control %s is sampling now", getCName());

        totVel_.clearDevice(stream);
        for (auto &pv : pvs_)
            _sampleOnePv(pv, stream);
        totVel_.downloadFromDevice(stream);
        accumulatedTotVel_.x += totVel_[0].x;
        accumulatedTotVel_.y += totVel_[0].y;
        accumulatedTotVel_.z += totVel_[0].z;
    }

    if (!isTimeEvery(getState(), tuneEvery_)) return;

    nSamples_.downloadFromDevice(stream);
    nSamples_.clearDevice(stream);

    long nSamplesTot = 0;
    double3 totVelTot = make_double3(0,0,0);

    const long nSamplesLoc = nSamples_[0];

    MPI_Check( MPI_Allreduce(&nSamplesLoc,         &nSamplesTot, 1, MPI_LONG,   MPI_SUM, comm_) );
    MPI_Check( MPI_Allreduce(&accumulatedTotVel_,  &totVelTot,   3, MPI_DOUBLE, MPI_SUM, comm_) );

    currentVel_ = nSamplesTot ? make_real3(totVelTot / nSamplesTot) : make_real3(0._r, 0._r, 0._r);
    force_ = pid_.update(targetVel_ - currentVel_);
    accumulatedTotVel_ = {0,0,0};
}

void SimulationVelocityControl::serializeAndSend(__UNUSED hipStream_t stream)
{
    if (!isTimeEvery(getState(), dumpEvery_)) return;

    _waitPrevSend();
    SimpleSerializer::serialize(sendBuffer_, getState()->currentTime, getState()->currentStep, currentVel_, force_);
    _send(sendBuffer_);
}

void SimulationVelocityControl::checkpoint(MPI_Comm comm, const std::string& path, int checkpointId)
{
    const auto filename = createCheckpointNameWithId(path, "plugin." + getName(), "txt", checkpointId);

    text_IO::write(filename, pid_);

    createCheckpointSymlink(comm, path, "plugin." + getName(), "txt", checkpointId);
}

void SimulationVelocityControl::restart(__UNUSED MPI_Comm comm, const std::string& path)
{
    const auto filename = createCheckpointName(path, "plugin." + getName(), "txt");
    const bool good = text_IO::read(filename, pid_);
    if (!good) die("failed to read '%s'\n", filename.c_str());
}




PostprocessVelocityControl::PostprocessVelocityControl(std::string name, std::string filename) :
    PostprocessPlugin(name)
{
    filename = setExtensionOrDie(filename, "csv");
    auto status = fdump_.open(filename, "w");
    if (status != FileWrapper::Status::Success)
        die("Could not open file '%s'", filename.c_str());
    fprintf(fdump_.get(), "time,time_step,vx,vy,vz,fx,fy,fz\n");
}

void PostprocessVelocityControl::deserialize()
{
    MirState::StepType currentTimeStep;
    MirState::TimeType currentTime;
    real3 vel, force;

    SimpleSerializer::deserialize(data_, currentTime, currentTimeStep, vel, force);

    if (rank_ == 0)
    {
        fprintf(fdump_.get(),
                "%g,%lld,"
                "%g,%g,%g,"
                "%g,%g,%g\n",
                currentTime, currentTimeStep,
                vel.x, vel.y, vel.z,
                force.x, force.y, force.z);

        fflush(fdump_.get());
    }
}

} // namespace mirheo
