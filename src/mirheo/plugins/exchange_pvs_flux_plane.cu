#include "hip/hip_runtime.h"
#include "exchange_pvs_flux_plane.h"

#include <mirheo/core/pvs/packers/particles.h>
#include <mirheo/core/pvs/particle_vector.h>
#include <mirheo/core/pvs/views/pv.h>
#include <mirheo/core/simulation.h>
#include <mirheo/core/utils/cuda_common.h>
#include <mirheo/core/utils/kernel_launch.h>

namespace mirheo
{

namespace exchange_pvs_flux_plane_kernels
{

__device__ inline bool sidePlane(real4 plane, real3 r)
{
    return plane.x * r.x + plane.y * r.y + plane.z * r.z + plane.w >= 0._r;
}

__device__ inline bool hasCrossedPlane(DomainInfo domain, real3 pos, real3 oldPos, real4 plane)
{
    pos    = domain.local2global(pos);
    oldPos = domain.local2global(oldPos);
    return sidePlane(plane, pos) && !sidePlane(plane, oldPos);
}

__global__ void countParticles(DomainInfo domain, PVviewWithOldParticles view1, real4 plane, int *numberCrossed)
{
    int pid = blockIdx.x * blockDim.x + threadIdx.x;
    if (pid >= view1.size) return;

    Particle p;
    view1.readPosition   (p,    pid);
    auto rOld = view1.readOldPosition(pid);

    if (p.isMarked()) return;

    if (hasCrossedPlane(domain, p.r, rOld, plane))
        atomicAdd(numberCrossed, 1);
}

__global__ void moveParticles(DomainInfo domain, PVviewWithOldParticles view1, PVview view2,
                              real4 plane, int oldsize2, int *numberCrossed,
                              ParticlePackerHandler extra1, ParticlePackerHandler extra2)
{
    int pid = blockIdx.x * blockDim.x + threadIdx.x;
    if (pid >= view1.size) return;

    Particle p;
    view1.readPosition(p, pid);
    auto rOld = view1.readOldPosition(pid);

    if (p.isMarked()) return;

    if (hasCrossedPlane(domain, p.r, rOld, plane))
    {
        int dst = atomicAdd(numberCrossed, 1);
        dst += oldsize2;

        view1.readVelocity(p, pid);
        view2.writeParticle(dst, p);

        p.mark();
        view1.writeParticle(pid, p);

        extra1.particles.copyTo(extra2.particles, pid, dst);
    }
}

} // namespace exchange_pvs_flux_plane_kernels


ExchangePVSFluxPlanePlugin::ExchangePVSFluxPlanePlugin(const MirState *state, std::string name, std::string pv1Name, std::string pv2Name, real4 plane) :
    SimulationPlugin(state, name),
    pv1Name_(pv1Name),
    pv2Name_(pv2Name),
    plane_(plane),
    numberCrossedParticles_(1)
{
    // we will copy positions and velocities manually in the kernel
    PackPredicate predicate = [](const DataManager::NamedChannelDesc& namedDesc)
    {
        auto channelName = namedDesc.first;
        auto channelDesc = namedDesc.second;
        return
            (channelName != channel_names::positions) &&
            (channelName != channel_names::velocities) &&
            (channelDesc->persistence == DataManager::PersistenceMode::Active);
    };

    extra1_ = std::make_unique<ParticlePacker>(predicate);
    extra2_ = std::make_unique<ParticlePacker>(predicate);
}

ExchangePVSFluxPlanePlugin::~ExchangePVSFluxPlanePlugin() = default;

void ExchangePVSFluxPlanePlugin::setup(Simulation* simulation, const MPI_Comm& comm, const MPI_Comm& interComm)
{
    SimulationPlugin::setup(simulation, comm, interComm);

    pv1_ = simulation->getPVbyNameOrDie(pv1Name_);
    pv2_ = simulation->getPVbyNameOrDie(pv2Name_);

    pv1_->requireDataPerParticle<real4> (channel_names::oldPositions, DataManager::PersistenceMode::Active, DataManager::ShiftMode::Active);
    pv2_->requireDataPerParticle<real4> (channel_names::oldPositions, DataManager::PersistenceMode::Active, DataManager::ShiftMode::Active);
}

void ExchangePVSFluxPlanePlugin::beforeCellLists(hipStream_t stream)
{
    const DomainInfo domain = getState()->domain;
    PVviewWithOldParticles view1(pv1_, pv1_->local());
    PVview                 view2(pv2_, pv2_->local());
    const int nthreads = 128;

    numberCrossedParticles_.clear(stream);

    SAFE_KERNEL_LAUNCH(
            exchange_pvs_flux_plane_kernels::countParticles,
            getNblocks(view1.size, nthreads), nthreads, 0, stream,
            domain, view1, plane_, numberCrossedParticles_.devPtr() );

    numberCrossedParticles_.downloadFromDevice(stream, ContainersSynch::Synch);

    const int numPartsExchange = numberCrossedParticles_[0];
    const int old_size2 = view2.size;
    const int new_size2 = old_size2 + numPartsExchange;

    pv2_->local()->resize(new_size2, stream);
    numberCrossedParticles_.clear(stream);

    view2 = PVview(pv2_, pv2_->local());

    extra1_->update(pv1_->local(), stream);
    extra2_->update(pv2_->local(), stream);

    SAFE_KERNEL_LAUNCH(
        exchange_pvs_flux_plane_kernels::moveParticles,
        getNblocks(view1.size, nthreads), nthreads, 0, stream,
        domain, view1, view2, plane_, old_size2, numberCrossedParticles_.devPtr(),
        extra1_->handler(), extra2_->handler() );

    if (numPartsExchange > 0)
    {
        pv1_->cellListStamp++;
        pv2_->cellListStamp++;
    }
}

} // namespace mirheo
