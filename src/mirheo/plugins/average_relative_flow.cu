#include "hip/hip_runtime.h"
// Copyright 2020 ETH Zurich. All Rights Reserved.
#include "average_relative_flow.h"

#include "utils/sampling_helpers.h"
#include "utils/simple_serializer.h"
#include "utils/time_stamp.h"

#include <mirheo/core/celllist.h>
#include <mirheo/core/pvs/object_vector.h>
#include <mirheo/core/pvs/views/pv.h>
#include <mirheo/core/simulation.h>
#include <mirheo/core/utils/cuda_common.h>
#include <mirheo/core/utils/kernel_launch.h>
#include <mirheo/core/utils/mpi_types.h>

namespace mirheo
{

namespace average_relative_flow_kernels
{
__global__ void sampleRelative(
        PVview pvView, CellListInfo cinfo,
        real* avgDensity,
        ChannelsInfo channelsInfo,
        real3 relativePoint)
{
    const int pid = threadIdx.x + blockIdx.x*blockDim.x;
    if (pid >= pvView.size) return;

    real3 r = make_real3(pvView.readPosition(pid));
    r -= relativePoint;

    int3 cid3 = cinfo.getCellIdAlongAxes<CellListsProjection::NoClamp>(r);
    cid3 = (cid3 + cinfo.ncells) % cinfo.ncells;
    const int cid = cinfo.encode(cid3);

    atomicAdd(avgDensity + cid, 1);

    sampling_helpers_kernels::sampleChannels(pid, cid, channelsInfo);
}
} // namespace average_relative_flow_kernels

AverageRelative3D::AverageRelative3D(
       const MirState *state, std::string name, std::vector<std::string> pvNames,
       std::vector<std::string> channelNames, int sampleEvery,
       int dumpEvery, real3 binSize, std::string relativeOVname, int relativeID) :
    Average3D(state, name, pvNames, channelNames, sampleEvery, dumpEvery, binSize),
    relativeOVname_(relativeOVname),
    relativeID_(relativeID)
{}

void AverageRelative3D::setup(Simulation* simulation, const MPI_Comm& comm, const MPI_Comm& interComm)
{
    Average3D::setup(simulation, comm, interComm);

    int local_size = numberDensity_.size();
    int global_size = local_size * nranks_;

    localNumberDensity_      .resize(local_size);
    numberDensity_           .resize_anew(global_size);
    accumulatedNumberDensity_.resize_anew(global_size);
    numberDensity_.clear(defaultStream);

    localChannels_.resize(channelsInfo_.n);

    for (int i = 0; i < channelsInfo_.n; ++i)
    {
        local_size  = channelsInfo_.average[i].size();
        global_size = local_size * nranks_;
        localChannels_[i].resize(local_size);
        channelsInfo_.average[i].resize_anew(global_size);
        accumulatedAverage_  [i].resize_anew(global_size);
        channelsInfo_.average[i].clear(defaultStream);
        channelsInfo_.averagePtrs[i] = channelsInfo_.average[i].devPtr();
    }

    channelsInfo_.averagePtrs.uploadToDevice(defaultStream);
    channelsInfo_.types.uploadToDevice(defaultStream);

    // Relative stuff
    relativeOV_ = simulation->getOVbyNameOrDie(relativeOVname_);

    if ( !relativeOV_->local()->dataPerObject.checkChannelExists(channel_names::motions) )
        die("Only rigid objects are supported for relative flow, but got OV '%s'", relativeOV_->getCName());

    const int locsize = relativeOV_->local()->getNumObjects();
    int totsize {0};

    MPI_Check( MPI_Reduce(&locsize, &totsize, 1, MPI_INT, MPI_SUM, 0, comm) );

    if (rank_ == 0 && relativeID_ >= totsize)
        die("Too few objects in OV '%s' (only %d); but requested id %d",
            relativeOV_->getCName(), totsize, relativeID_);
}

void AverageRelative3D::sampleOnePv(real3 relativeParam, ParticleVector *pv, hipStream_t stream)
{
    const CellListInfo cinfo(binSize_, getState()->domain.globalSize);
    PVview pvView(pv, pv->local());
    ChannelsInfo gpuInfo(channelsInfo_, pv, stream);

    const int nthreads = 128;
    SAFE_KERNEL_LAUNCH
        (average_relative_flow_kernels::sampleRelative,
         getNblocks(pvView.size, nthreads), nthreads, 0, stream,
         pvView, cinfo, numberDensity_.devPtr(), gpuInfo, relativeParam);
}

void AverageRelative3D::afterIntegration(hipStream_t stream)
{
    const int TAG = 22;
    const int NCOMPONENTS = 2 * sizeof(real3) / sizeof(real);

    if (!isTimeEvery(getState(), sampleEvery_)) return;

    debug2("Plugin %s is sampling now", getCName());

    real3 relativeParams[2] = {make_real3(0.0_r), make_real3(0.0_r)};

    // Find and broadcast the position and velocity of the relative object
    MPI_Request req;
    MPI_Check( MPI_Irecv(relativeParams, NCOMPONENTS, getMPIFloatType<real>(), MPI_ANY_SOURCE, TAG, comm_, &req) );

    auto ids     = relativeOV_->local()->dataPerObject.getData<int64_t>(channel_names::globalIds);
    auto motions = relativeOV_->local()->dataPerObject.getData<RigidMotion>(channel_names::motions);

    ids    ->downloadFromDevice(stream, ContainersSynch::Asynch);
    motions->downloadFromDevice(stream, ContainersSynch::Synch);

    for (size_t i = 0; i < ids->size(); i++)
    {
        if ((*ids)[i] == relativeID_)
        {
            real3 params[2] = { make_real3( (*motions)[i].r   ),
                                make_real3( (*motions)[i].vel ) };

            params[0] = getState()->domain.local2global(params[0]);

            for (int r = 0; r < nranks_; r++)
                MPI_Send(&params, NCOMPONENTS, getMPIFloatType<real>(), r, TAG, comm_);

            break;
        }
    }

    MPI_Check( MPI_Wait(&req, MPI_STATUS_IGNORE) );

    relativeParams[0] = getState()->domain.global2local(relativeParams[0]);

    for (auto& pv : pvs_)
        sampleOnePv(relativeParams[0], pv, stream);

    accumulateSampledAndClear(stream);

    averageRelativeVelocity_ += relativeParams[1];

    nSamples_++;
}


void AverageRelative3D::extractLocalBlock()
{
    static const double scale_by_density = -1.0;

    auto oneChannel = [this] (const PinnedBuffer<double>& channel, Average3D::ChannelType type, double scale, std::vector<double>& dest) {

        MPI_Check( MPI_Allreduce(MPI_IN_PLACE, channel.hostPtr(), channel.size(), MPI_DOUBLE, MPI_SUM, comm_) );

        const int ncomponents = this->getNcomponents(type);

        const int3 globalResolution = resolution_ * nranks3D_;

        double factor;
        int dstId = 0;
        for (int k = rank3D_.z * resolution_.z; k < (rank3D_.z+1) * resolution_.z; ++k)
        {
            for (int j = rank3D_.y * resolution_.y; j < (rank3D_.y+1) * resolution_.y; ++j)
            {
                for (int i = rank3D_.x * resolution_.x; i < (rank3D_.x+1) * resolution_.x; ++i)
                {
                    const int scalId = (k*globalResolution.y*globalResolution.x + j*globalResolution.x + i);
                    int srcId = ncomponents * scalId;
                    for (int c = 0; c < ncomponents; ++c)
                    {
                        if (scale == scale_by_density) factor = 1.0_r / accumulatedNumberDensity_[scalId];
                        else                           factor = scale;

                        dest[dstId++] = channel[srcId] * factor;
                        srcId++;
                    }
                }
            }
        }
    };

    // Order is important! Density comes first
    oneChannel(accumulatedNumberDensity_, Average3D::ChannelType::Scalar, 1.0 / (nSamples_ * binSize_.x*binSize_.y*binSize_.z), localNumberDensity_);

    for (int i = 0; i < channelsInfo_.n; ++i)
        oneChannel(accumulatedAverage_[i], channelsInfo_.types[i], scale_by_density, localChannels_[i]);
}

void AverageRelative3D::serializeAndSend(hipStream_t stream)
{
    if (!isTimeEvery(getState(), dumpEvery_)) return;

    for (int i = 0; i < channelsInfo_.n; ++i)
    {
        auto& data = accumulatedAverage_[i];

        if (channelsInfo_.names[i] == channel_names::velocities)
        {
            constexpr int nthreads = 128;
            const int numVec3 = data.size() / 3;

            SAFE_KERNEL_LAUNCH
                (sampling_helpers_kernels::correctVelocity,
                 getNblocks(numVec3, nthreads), nthreads, 0, stream,
                 numVec3, reinterpret_cast<double3*> (data.devPtr()),
                 accumulatedNumberDensity_.devPtr(), averageRelativeVelocity_ / static_cast<real>(nSamples_));

            averageRelativeVelocity_ = make_real3(0.0_r);
        }
    }

    accumulatedNumberDensity_.downloadFromDevice(stream, ContainersSynch::Asynch);
    accumulatedNumberDensity_.clearDevice(stream);

    for (auto& data : accumulatedAverage_)
    {
        data.downloadFromDevice(stream, ContainersSynch::Asynch);
        data.clearDevice(stream);
    }

    CUDA_Check( hipStreamSynchronize(stream) );

    extractLocalBlock();
    nSamples_ = 0;

    MirState::StepType timeStamp = getTimeStamp(getState(), dumpEvery_) - 1; // -1 to start from 0

    debug2("Plugin '%s' is now packing the data", getCName());
    _waitPrevSend();
    SimpleSerializer::serialize(sendBuffer_, getState()->currentTime, timeStamp, localNumberDensity_, localChannels_);
    _send(sendBuffer_);
}

} // namespace mirheo
