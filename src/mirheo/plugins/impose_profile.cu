#include "hip/hip_runtime.h"
// Copyright 2020 ETH Zurich. All Rights Reserved.
#include "impose_profile.h"

#include <mirheo/core/celllist.h>
#include <mirheo/core/pvs/particle_vector.h>
#include <mirheo/core/pvs/views/pv.h>
#include <mirheo/core/simulation.h>
#include <mirheo/core/utils/cuda_common.h>
#include <mirheo/core/utils/cuda_rng.h>
#include <mirheo/core/utils/kernel_launch.h>

namespace mirheo
{

__device__ inline bool all_lt(real3 a, real3 b)
{
    return a.x < b.x && a.y < b.y && a.z < b.z;
}

__global__ void applyProfile(
        CellListInfo cinfo, PVview view,
        const int* relevantCells, const int nRelevantCells,
        real3 low, real3 high,
        real3 targetVel,
        real kBT, real invMass, real seed1, real seed2)
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid >= nRelevantCells) return;

    int pstart = cinfo.cellStarts[ relevantCells[gid]   ];
    int pend   = cinfo.cellStarts[ relevantCells[gid]+1 ];

#pragma unroll 3
    for (int pid = pstart; pid < pend; pid++)
    {
        Particle p(view.readParticle(pid));

        if (all_lt(low, p.r) && all_lt(p.r, high))
        {
            real2 rand1 = Saru::normal2(seed1 + pid, threadIdx.x, blockIdx.x);
            real2 rand2 = Saru::normal2(seed2 + pid, threadIdx.x, blockIdx.x);

            p.u = targetVel + math::sqrt(kBT * invMass) * make_real3(rand1.x, rand1.y, rand2.x);
            view.writeParticle(pid, p);
        }
    }
}

template<bool QUERY>
__global__ void getRelevantCells(
        CellListInfo cinfo,
        real3 low, real3 high,
        int* relevantCells, int* nRelevantCells)
{
    const int cid = blockIdx.x * blockDim.x + threadIdx.x;
    if (cid >= cinfo.totcells) return;

    int3 ind;
    cinfo.decode(cid, ind.x, ind.y, ind.z);
    real3 botCell = -0.5_r * cinfo.localDomainSize + make_real3(ind)*cinfo.h;
    real3 topCell = botCell + cinfo.h;

    bool relevant = all_lt(low, topCell) && all_lt(botCell, high);

    if (relevant)
    {
        int id = atomicAggInc(nRelevantCells);
        if (!QUERY) relevantCells[id] = cid;
    }
}

ImposeProfilePlugin::ImposeProfilePlugin(const MirState *state, std::string name, std::string pvName,
                                         real3 low, real3 high, real3 targetVel, real kBT) :
    SimulationPlugin(state, name),
    pvName_(pvName),
    low_(low),
    high_(high),
    targetVel_(targetVel),
    kBT_(kBT)
{}

void ImposeProfilePlugin::setup(Simulation* simulation, const MPI_Comm& comm, const MPI_Comm& interComm)
{
    SimulationPlugin::setup(simulation, comm, interComm);

    pv_ = simulation->getPVbyNameOrDie(pvName_);
    cl_ = simulation->gelCellList(pv_);

    if (cl_ == nullptr)
        die("Cell-list is required for PV '%s' by plugin '%s'", pvName_.c_str(), getCName());

    debug("Setting up pluging '%s' to impose uniform profile with velocity [%f %f %f]"
          " and temperature %f in a box [%.2f %.2f %.2f] - [%.2f %.2f %.2f] for PV '%s'",
          getCName(), targetVel_.x, targetVel_.y, targetVel_.z, kBT_,
          low_.x, low_.y, low_.z, high_.x, high_.y, high_.z, pv_->getCName());

    low_  = getState()->domain.global2local(low_);
    high_ = getState()->domain.global2local(high_);

    const int nthreads = 128;

    nRelevantCells_.clearDevice(defaultStream);
    SAFE_KERNEL_LAUNCH(
            getRelevantCells<true>,
            getNblocks(cl_->totcells, nthreads), nthreads, 0, defaultStream,
            cl_->cellInfo(), low_, high_, relevantCells_.devPtr(), nRelevantCells_.devPtr() );

    nRelevantCells_.downloadFromDevice(defaultStream);
    relevantCells_.resize_anew(nRelevantCells_[0]);
    nRelevantCells_.clearDevice(defaultStream);

    SAFE_KERNEL_LAUNCH(
            getRelevantCells<false>,
            getNblocks(cl_->totcells, nthreads), nthreads, 0, defaultStream,
            cl_->cellInfo(), low_, high_, relevantCells_.devPtr(), nRelevantCells_.devPtr() );
}

void ImposeProfilePlugin::afterIntegration(hipStream_t stream)
{
    const int nthreads = 128;

    debug2("Imposing uniform profile for PV '%s' as per plugin '%s'",
           pv_->getCName(), getCName());

    SAFE_KERNEL_LAUNCH(
            applyProfile,
            getNblocks(nRelevantCells_[0], nthreads), nthreads, 0, stream,
            cl_->cellInfo(), cl_->getView<PVview>(), relevantCells_.devPtr(), nRelevantCells_[0], low_, high_, targetVel_,
            kBT_, 1.0_r / pv_->getMassPerParticle(), drand48(), drand48() );
}

} // namespace mirheo
