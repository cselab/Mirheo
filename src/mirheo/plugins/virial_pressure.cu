#include "hip/hip_runtime.h"
// Copyright 2020 ETH Zurich. All Rights Reserved.
#include "virial_pressure.h"
#include "utils/simple_serializer.h"
#include "utils/time_stamp.h"

#include <mirheo/core/datatypes.h>
#include <mirheo/core/pvs/particle_vector.h>
#include <mirheo/core/pvs/views/pv.h>
#include <mirheo/core/simulation.h>
#include <mirheo/core/utils/path.h>
#include <mirheo/core/utils/common.h>
#include <mirheo/core/utils/cuda_common.h>
#include <mirheo/core/utils/kernel_launch.h>
#include <mirheo/core/utils/mpi_types.h>

namespace mirheo
{

namespace virial_pressure_kernels
{
__global__ void totalPressure(PVview view, const Stress *stress, FieldDeviceHandler region, virial_pressure_plugin::ReductionType *pressure)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    virial_pressure_plugin::ReductionType P = 0;
    Particle p;

    if (tid < view.size) {
        const Stress s = stress[tid];
        auto r = Real3_int(view.readPosition(tid)).v;

        if (region(r) > 0)
            P = (s.xx + s.yy + s.zz) / 3.0;
    }

    P = warpReduce(P, [](virial_pressure_plugin::ReductionType a, virial_pressure_plugin::ReductionType b) { return a+b; });

    if (laneId() == 0)
        atomicAdd(pressure, P);
}
} // namespace virial_pressure_kernels

VirialPressurePlugin::VirialPressurePlugin(const MirState *state, std::string name, std::string pvName,
                                           FieldFunction func, real3 h, int dumpEvery) :
    SimulationPlugin(state, name),
    pvName_(pvName),
    dumpEvery_(dumpEvery),
    region_(state, "field_"+name, func, h)
{}

VirialPressurePlugin::~VirialPressurePlugin() = default;

void VirialPressurePlugin::setup(Simulation* simulation, const MPI_Comm& comm, const MPI_Comm& interComm)
{
    SimulationPlugin::setup(simulation, comm, interComm);

    pv_ = simulation->getPVbyNameOrDie(pvName_);

    region_.setup(comm);

    info("Plugin %s initialized for the following particle vector: %s", getCName(), pvName_.c_str());
}

void VirialPressurePlugin::handshake()
{
    SimpleSerializer::serialize(sendBuffer_, pvName_);
    _send(sendBuffer_);
}

void VirialPressurePlugin::afterIntegration(hipStream_t stream)
{
    if (!isTimeEvery(getState(), dumpEvery_)) return;

    PVview view(pv_, pv_->local());
    const Stress *stress = pv_->local()->dataPerParticle.getData<Stress>(channel_names::stresses)->devPtr();

    localVirialPressure_.clear(stream);

    constexpr int nthreads = 128;
    const int nblocks = getNblocks(view.size, nthreads);

    SAFE_KERNEL_LAUNCH(
        virial_pressure_kernels::totalPressure,
        nblocks, nthreads, 0, stream,
        view, stress, region_.handler(), localVirialPressure_.devPtr() );

    localVirialPressure_.downloadFromDevice(stream, ContainersSynch::Synch);

    savedTime_ = getState()->currentTime;
    needToSend_ = true;
}

void VirialPressurePlugin::serializeAndSend(__UNUSED hipStream_t stream)
{
    if (!needToSend_) return;

    debug2("Plugin %s is sending now data", getCName());

    _waitPrevSend();
    SimpleSerializer::serialize(sendBuffer_, savedTime_, localVirialPressure_[0]);
    _send(sendBuffer_);

    needToSend_ = false;
}

//=================================================================================

VirialPressureDumper::VirialPressureDumper(std::string name, std::string path) :
    PostprocessPlugin(name),
    path_(makePath(path))
{}

void VirialPressureDumper::setup(const MPI_Comm& comm, const MPI_Comm& interComm)
{
    PostprocessPlugin::setup(comm, interComm);
    activated_ = createFoldersCollective(comm, path_);
}

void VirialPressureDumper::handshake()
{
    auto req = waitData();
    MPI_Check( MPI_Wait(&req, MPI_STATUS_IGNORE) );
    recv();

    std::string pvName;
    SimpleSerializer::deserialize(data_, pvName);

    if (activated_ && fdump_.get() == nullptr)
    {
        auto fname = joinPaths(path_, setExtensionOrDie(pvName, "csv"));
        auto status = fdump_.open(fname, "w");
        if (status != FileWrapper::Status::Success)
            die("Could not open file '%s'", fname.c_str());
        fprintf(fdump_.get(), "time,pressure\n");
    }
}

void VirialPressureDumper::deserialize()
{
    MirState::TimeType curTime;
    virial_pressure_plugin::ReductionType localPressure, totalPressure;

    SimpleSerializer::deserialize(data_, curTime, localPressure);

    if (!activated_) return;

    const auto dataType = getMPIFloatType<virial_pressure_plugin::ReductionType>();
    MPI_Check( MPI_Reduce(&localPressure, &totalPressure, 1, dataType, MPI_SUM, 0, comm_) );

    fprintf(fdump_.get(), "%g,%.6e\n", curTime, totalPressure);
}

} // namespace mirheo
