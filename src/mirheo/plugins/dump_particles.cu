#include "hip/hip_runtime.h"
// Copyright 2020 ETH Zurich. All Rights Reserved.
#include "dump_particles.h"
#include "utils/simple_serializer.h"
#include "utils/time_stamp.h"

#include <mirheo/core/pvs/rod_vector.h>
#include <mirheo/core/simulation.h>
#include <mirheo/core/utils/config.h>
#include <mirheo/core/utils/cuda_common.h>
#include <mirheo/core/utils/path.h>
#include <mirheo/core/utils/kernel_launch.h>
#include <mirheo/core/xdmf/type_map.h>

namespace mirheo
{

namespace dump_particles_kernels
{

template <typename T>
__global__ void copyObjectDataToParticles(int objSize, int nObjects, const T *srcObjData, T *dstParticleData)
{
    const int pid   = threadIdx.x + blockIdx.x * blockDim.x;
    const int objId = pid / objSize;

    if (objId >= nObjects) return;

    dstParticleData[pid] = srcObjData[objId];
}

template <typename T>
__global__ void copyRodDataToParticles(int numBiSegmentsPerObject, int objSize, int nObjects, const T *rodData, T *particleData)
{
    constexpr int stride = 5;
    const int pid = threadIdx.x + blockIdx.x * blockDim.x;

    const int objId        = pid / objSize;
    const int localPartId  = pid % objSize;
    const int localBisegId = math::min(localPartId / stride, numBiSegmentsPerObject); // min because of last particle

    const int bid = objId * numBiSegmentsPerObject + localBisegId;

    if (objId < nObjects)
        particleData[pid] = rodData[bid];
}

} // namespace dump_particles_kernels


ParticleSenderPlugin::ParticleSenderPlugin(const MirState *state, std::string name, std::string pvName, int dumpEvery,
                                           const std::vector<std::string>& channelNames) :
    SimulationPlugin(state, name),
    pvName_(pvName),
    dumpEvery_(dumpEvery),
    channelNames_(channelNames)
{
    channelData_.resize(channelNames_.size());
}

ParticleSenderPlugin::ParticleSenderPlugin(const MirState *state, Loader& loader, const ConfigObject& config) :
    ParticleSenderPlugin(state, config["name"], config["pvName"], config["dumpEvery"],
                         loader.load<std::vector<std::string>>(config["channelNames"]))
{}

ParticleSenderPlugin::~ParticleSenderPlugin() = default;

void ParticleSenderPlugin::setup(Simulation *simulation, const MPI_Comm& comm, const MPI_Comm& interComm)
{
    SimulationPlugin::setup(simulation, comm, interComm);

    pv_ = simulation->getPVbyNameOrDie(pvName_);

    info("Plugin %s initialized for the following particle vector: %s", getCName(), pvName_.c_str());
}

void ParticleSenderPlugin::handshake()
{
    std::vector<XDMF::Channel::DataForm> dataForms;
    std::vector<XDMF::Channel::NumberType> numberTypes;
    std::vector<std::string> typeDescriptorsStr;

    auto pushChannelInfos = [&dataForms, &numberTypes, &typeDescriptorsStr](const DataManager::ChannelDescription& desc)
    {
        mpark::visit([&dataForms, &numberTypes, &typeDescriptorsStr](auto pinnedBufferPtr)
        {
            using T = typename std::remove_pointer<decltype(pinnedBufferPtr)>::type::value_type;
            dataForms         .push_back(XDMF::getDataForm  <T>());
            numberTypes       .push_back(XDMF::getNumberType<T>());
            typeDescriptorsStr.push_back(typeDescriptorToString(DataTypeWrapper<T>{}));
        }, desc.varDataPtr);
    };

    auto ov = dynamic_cast<ObjectVector*>(pv_);
    auto rv = dynamic_cast<RodVector*>(pv_);

    for (const auto& name : channelNames_)
    {
        if (pv_->local()->dataPerParticle.checkChannelExists(name))
        {
            const auto& desc = pv_->local()->dataPerParticle.getChannelDescOrDie(name);
            pushChannelInfos(desc);
        }
        else if (ov != nullptr && ov->local()->dataPerObject.checkChannelExists(name))
        {
            const auto& desc = ov->local()->dataPerObject.getChannelDescOrDie(name);
            pushChannelInfos(desc);
        }
        else if (rv != nullptr && rv->local()->dataPerBisegment.checkChannelExists(name))
        {
            const auto& desc = rv->local()->dataPerBisegment.getChannelDescOrDie(name);
            pushChannelInfos(desc);
        }
        else
        {
            die("Channel not found: '%s' in particle vector '%s'",
                getCName(), pv_->getCName());
        }
    }

    _waitPrevSend();
    SimpleSerializer::serialize(sendBuffer_, channelNames_, dataForms, numberTypes, typeDescriptorsStr);
    _send(sendBuffer_);
}

static inline void copyData(ParticleVector *pv, const std::string& channelName, HostBuffer<char>& dst, hipStream_t stream)
{
    auto srcContainer = pv->local()->dataPerParticle.getGenericData(channelName);
    dst.genericCopy(srcContainer, stream);
}

static inline void copyData(ObjectVector *ov, const std::string& channelName, HostBuffer<char>& dst, DeviceBuffer<char>& workSpace, hipStream_t stream)
{
    auto lov = ov->local();

    const auto& srcDesc = lov->dataPerObject.getChannelDescOrDie(channelName);

    const int objSize  = lov->getObjectSize();
    const int nObjects = lov->getNumObjects();

    mpark::visit([&](auto srcBufferPtr)
    {
        using T = typename std::remove_pointer<decltype(srcBufferPtr)>::type::value_type;

        constexpr int nthreads = 128;
        const int nParts = objSize * nObjects;
        const int nblocks = getNblocks(nParts, nthreads);

        workSpace.resize_anew(nParts * sizeof(T));

        SAFE_KERNEL_LAUNCH(
            dump_particles_kernels::copyObjectDataToParticles,
            nblocks, nthreads, 0, stream,
            objSize, nObjects, srcBufferPtr->devPtr(),
            reinterpret_cast<T*>(workSpace.devPtr()));
    }, srcDesc.varDataPtr);

    dst.genericCopy(&workSpace, stream);
}

static inline void copyData(RodVector *rv, const std::string& channelName, HostBuffer<char>& dst, DeviceBuffer<char>& workSpace, hipStream_t stream)
{
    auto lrv = rv->local();

    const auto& srcDesc = lrv->dataPerBisegment.getChannelDescOrDie(channelName);

    const int objSize  = lrv->getObjectSize();
    const int nObjects = lrv->getNumObjects();
    const int numBiSegmentsPerObject = lrv->getNumSegmentsPerRod() - 1;

    mpark::visit([&](auto srcBufferPtr)
    {
        using T = typename std::remove_pointer<decltype(srcBufferPtr)>::type::value_type;

        constexpr int nthreads = 128;
        const int nParts = objSize * nObjects;
        const int nblocks = getNblocks(nParts, nthreads);

        workSpace.resize_anew(nParts * sizeof(T));

        SAFE_KERNEL_LAUNCH(
            dump_particles_kernels::copyRodDataToParticles,
            nblocks, nthreads, 0, stream,
            numBiSegmentsPerObject, objSize, nObjects, srcBufferPtr->devPtr(),
            reinterpret_cast<T*>(workSpace.devPtr()));
    }, srcDesc.varDataPtr);

    dst.genericCopy(&workSpace, stream);
}

void ParticleSenderPlugin::beforeForces(hipStream_t stream)
{
    if (!isTimeEvery(getState(), dumpEvery_)) return;

    positions_ .genericCopy(&pv_->local()->positions() , stream);
    velocities_.genericCopy(&pv_->local()->velocities(), stream);

    auto ov = dynamic_cast<ObjectVector*>(pv_);
    auto rv = dynamic_cast<RodVector*>(pv_);

    for (size_t i = 0; i < channelNames_.size(); ++i)
    {
        auto name = channelNames_[i];

        if (pv_->local()->dataPerParticle.checkChannelExists(name))
        {
            copyData(pv_, name, channelData_[i], stream);
        }
        else if (ov != nullptr && ov->local()->dataPerObject.checkChannelExists(name))
        {
            copyData(ov, name, channelData_[i], workSpace_, stream);
        }
        else if (rv != nullptr && rv->local()->dataPerBisegment.checkChannelExists(name))
        {
            copyData(rv, name, channelData_[i], workSpace_, stream);
        }
        else
        {
            die("Channel not found: '%s' in particle vector '%s'",
                getCName(), pv_->getCName());
        }
    }
}

void ParticleSenderPlugin::serializeAndSend(__UNUSED hipStream_t stream)
{
    if (!isTimeEvery(getState(), dumpEvery_)) return;

    debug2("Plugin %s is sending now data", getCName());

    for (auto& p : positions_)
    {
        auto r = getState()->domain.local2global(make_real3(p));
        p.x = r.x; p.y = r.y; p.z = r.z;
    }

    const MirState::StepType timeStamp = getTimeStamp(getState(), dumpEvery_);

    debug2("Plugin %s is packing now data consisting of %zu particles",
           getCName(), positions_.size());
    _waitPrevSend();
    SimpleSerializer::serialize(sendBuffer_, timeStamp, getState()->currentTime, positions_, velocities_, channelData_);
    _send(sendBuffer_);
}

void ParticleSenderPlugin::saveSnapshotAndRegister(Saver& saver)
{
    saver.registerObject(this, _saveSnapshot(saver, "ParticleSenderPlugin"));
}

ConfigObject ParticleSenderPlugin::_saveSnapshot(Saver& saver, const std::string& typeName)
{
    ConfigObject config = SimulationPlugin::_saveSnapshot(saver, typeName);
    config.emplace("pvName",       saver(pvName_));
    config.emplace("dumpEvery",    saver(dumpEvery_));
    config.emplace("channelNames", saver(channelNames_));
    return config;
}



ParticleDumperPlugin::ParticleDumperPlugin(std::string name, std::string path) :
    PostprocessPlugin(name),
    path_(path),
    positions_(std::make_shared<std::vector<real3>>())
{}

ParticleDumperPlugin::ParticleDumperPlugin(Loader&, const ConfigObject& config) :
    ParticleDumperPlugin(config["name"], config["path"])
{}

ParticleDumperPlugin::~ParticleDumperPlugin() = default;

void ParticleDumperPlugin::handshake()
{
    auto req = waitData();
    MPI_Check( MPI_Wait(&req, MPI_STATUS_IGNORE) );
    recv();

    std::vector<std::string> names;
    std::vector<XDMF::Channel::DataForm> dataForms;
    std::vector<XDMF::Channel::NumberType> numberTypes;
    std::vector<std::string> typeDescriptorsStr;

    SimpleSerializer::deserialize(data_, names, dataForms, numberTypes, typeDescriptorsStr);

    auto initChannel = [] (const std::string& name, XDMF::Channel::DataForm dataForm,
                           XDMF::Channel::NumberType numberType, TypeDescriptor datatype,
                           XDMF::Channel::NeedShift needShift = XDMF::Channel::NeedShift::False)
    {
        return XDMF::Channel{name, nullptr, dataForm, numberType, datatype, needShift};
    };

    // Velocity and id are special channels which are always present
    std::string allNames = "'velocity', 'id'";
    channels_.clear();
    channels_.push_back(initChannel("velocity", XDMF::Channel::DataForm::Vector, XDMF::getNumberType<real>(), DataTypeWrapper<real>()));
    channels_.push_back(initChannel("id",       XDMF::Channel::DataForm::Scalar, XDMF::Channel::NumberType::Int64, DataTypeWrapper<int64_t>()));

    for (size_t i = 0; i < names.size(); ++i)
    {
        const std::string& name = names[i];
        const auto dataForm   = dataForms[i];
        const auto numberType = numberTypes[i];
        const auto dataType   = stringToTypeDescriptor(typeDescriptorsStr[i]);

        const auto channel = initChannel(name, dataForm, numberType, dataType);

        channels_.push_back(channel);
        allNames += ", '" + name + "'";
    }

    // Create the required folder
    createFoldersCollective(comm_, getParentPath(path_));

    debug2("Plugin '%s' was set up to dump channels %s. Path is %s",
           getCName(), allNames.c_str(), path_.c_str());
}

static void unpackParticles(const std::vector<real4> &pos4, const std::vector<real4> &vel4,
                            std::vector<real3> &pos, std::vector<real3> &vel, std::vector<int64_t> &ids)
{
    const size_t n = pos4.size();
    pos.resize(n);
    vel.resize(n);
    ids.resize(n);

    for (size_t i = 0; i < n; ++i)
    {
        auto p = Particle(pos4[i], vel4[i]);
        pos[i] = p.r;
        vel[i] = p.u;
        ids[i] = p.getId();
    }
}

void ParticleDumperPlugin::_recvAndUnpack(MirState::TimeType &time, MirState::StepType& timeStamp)
{
    int c = 0;
    SimpleSerializer::deserialize(data_, timeStamp, time, pos4_, vel4_, channelData_);

    unpackParticles(pos4_, vel4_, *positions_, velocities_, ids_);

    channels_[c++].data = velocities_.data();
    channels_[c++].data = ids_.data();

    for (auto& cd : channelData_)
        channels_[c++].data = cd.data();
}

void ParticleDumperPlugin::deserialize()
{
    debug2("Plugin '%s' will dump right now", getCName());

    MirState::TimeType time;
    MirState::StepType timeStamp;
    _recvAndUnpack(time, timeStamp);

    std::string fname = path_ + createStrZeroPadded(timeStamp, zeroPadding_);

    XDMF::VertexGrid grid(positions_, comm_);
    XDMF::write(fname, &grid, channels_, time, comm_);
}

void ParticleDumperPlugin::saveSnapshotAndRegister(Saver& saver)
{
    saver.registerObject(this, _saveSnapshot(saver, "ParticleDumperPlugin"));
}

ConfigObject ParticleDumperPlugin::_saveSnapshot(Saver& saver, const std::string& typeName)
{
    ConfigObject config = PostprocessPlugin::_saveSnapshot(saver, typeName);
    config.emplace("path", saver(path_));
    return config;
}

} // namespace mirheo
