#include "hip/hip_runtime.h"
#include "dump_obj_stats.h"
#include "utils/simple_serializer.h"
#include "utils/time_stamp.h"

#include <mirheo/core/pvs/rigid_object_vector.h>
#include <mirheo/core/pvs/views/ov.h>
#include <mirheo/core/simulation.h>
#include <mirheo/core/utils/folders.h>
#include <mirheo/core/utils/hip/hip_vector_types.h>
#include <mirheo/core/utils/kernel_launch.h>

#include <iomanip>

namespace mirheo
{

namespace obj_stats_plugin_kernels
{

__global__ void collectObjStats(OVview view, RigidMotion *motionStats)
{
    const int objId  = blockIdx.x;
    const int tid    = threadIdx.x;
    const int laneId = tid % warpSize;

    RigidMotion local = {0};

    const real3 com = view.comAndExtents[objId].com;

    for (int i = tid; i < view.objSize; i += blockDim.x)
    {
        const int pid = objId * view.objSize + i;
        const Particle p = view.readParticle(pid);
        const real3 f = make_real3(view.forces[pid]);

        const real3 dr = p.r - com;

        local.vel    += p.u;
        local.omega  += cross(dr, p.u);
        local.force  += f;
        local.torque += cross(dr, f);
    }

    auto add = [](const RigidReal& a, const RigidReal& b) {return a+b;};

    warpReduce(local.vel,    add);
    warpReduce(local.omega,  add);
    warpReduce(local.force,  add);
    warpReduce(local.torque, add);

    if (laneId == 0)
    {
        atomicAdd( &motionStats[objId].vel,   local.vel   / view.objSize);
        atomicAdd( &motionStats[objId].omega, local.omega / view.objSize);

        atomicAdd( &motionStats[objId].force,  local.force );
        atomicAdd( &motionStats[objId].torque, local.torque);
    }
}

} // namespace obj_stats_plugin_kernels

ObjStatsPlugin::ObjStatsPlugin(const MirState *state, std::string name, std::string ovName, int dumpEvery) :
    SimulationPlugin(state, name),
    ovName_(ovName),
    dumpEvery_(dumpEvery)
{}

void ObjStatsPlugin::setup(Simulation *simulation, const MPI_Comm& comm, const MPI_Comm& interComm)
{
    SimulationPlugin::setup(simulation, comm, interComm);
    ov_ = simulation->getOVbyNameOrDie(ovName_);
    info("Plugin '%s' initialized for object vector '%s'", getCName(), ovName_.c_str());
}

void ObjStatsPlugin::handshake()
{
    SimpleSerializer::serialize(sendBuffer_, ovName_);
    _send(sendBuffer_);
}

void ObjStatsPlugin::afterIntegration(hipStream_t stream)
{
    if (!isTimeEvery(getState(), dumpEvery_)) return;

    auto lov = ov_->local();

    ids_ .copy( *lov->dataPerObject.getData<int64_t>     (channel_names::globalIds),  stream );
    coms_.copy( *lov->dataPerObject.getData<COMandExtent>(channel_names::comExtents), stream );

    if (auto rov = dynamic_cast<RigidObjectVector*>(ov_))
    {
        auto& oldMotions = *rov->local()->dataPerObject.getData<RigidMotion> (channel_names::oldMotions);
        motions_.copy(oldMotions, stream);
        isRov_ = true;
    }
    else
    {
        const int nthreads = 128;
        OVview view(ov_, lov);
        motionStats_.resize_anew(view.nObjects);

        motionStats_.clear(stream);

        SAFE_KERNEL_LAUNCH(
            obj_stats_plugin_kernels::collectObjStats,
            view.nObjects, nthreads, 0, stream,
            view, motionStats_.devPtr());

        motions_.copy(motionStats_, stream);
        isRov_ = false;
    }

    if (lov->dataPerObject.checkChannelExists(channel_names::membraneTypeId))
    {
        typeIds_.copy( *lov->dataPerObject.getData<int>(channel_names::membraneTypeId), stream);
        hasTypeIds_ = true;
    }

    savedTime_ = getState()->currentTime;
    needToSend_ = true;
}

void ObjStatsPlugin::serializeAndSend(__UNUSED hipStream_t stream)
{
    if (!needToSend_) return;

    debug2("Plugin %s is sending now data", getCName());

    _waitPrevSend();
    SimpleSerializer::serialize(sendBuffer_, savedTime_, getState()->domain, isRov_, ids_, coms_, motions_, hasTypeIds_, typeIds_);
    _send(sendBuffer_);

    needToSend_=false;
}

//=================================================================================

static void writeStats(MPI_Comm comm, DomainInfo domain, MPI_File& fout, real curTime, const std::vector<int64_t>& ids,
                       const std::vector<COMandExtent>& coms, const std::vector<RigidMotion>& motions, bool isRov,
                       bool hasTypeIds, const std::vector<int>& typeIds)
{
    const int np = ids.size();

    std::stringstream ss;
    ss.setf(std::ios::fixed, std::ios::floatfield);
    ss.precision(5);

    for (int i = 0; i < np; ++i)
    {
        auto com = coms[i].com;
        com = domain.local2global(com);

        ss << ids[i] << " " << curTime << "   "
           << std::setw(10) << com.x << " "
           << std::setw(10) << com.y << " "
           << std::setw(10) << com.z;

        const auto& motion = motions[i];

        if (isRov)
        {
            ss << "    "
               << std::setw(10) << motion.q.w << " "
               << std::setw(10) << motion.q.x << " "
               << std::setw(10) << motion.q.y << " "
               << std::setw(10) << motion.q.z;
        }

        ss << "    "
           << std::setw(10) << motion.vel.x << " "
           << std::setw(10) << motion.vel.y << " "
           << std::setw(10) << motion.vel.z << "    "

           << std::setw(10) << motion.omega.x << " "
           << std::setw(10) << motion.omega.y << " "
           << std::setw(10) << motion.omega.z << "    "

           << std::setw(10) << motion.force.x << " "
           << std::setw(10) << motion.force.y << " "
           << std::setw(10) << motion.force.z << "    "

           << std::setw(10) << motion.torque.x << " "
           << std::setw(10) << motion.torque.y << " "
           << std::setw(10) << motion.torque.z;

        if (hasTypeIds)
            ss << "    "  << typeIds[i];

        ss << std::endl;
    }

    const std::string content = ss.str();

    MPI_Offset offset = 0, size;
    MPI_Check( MPI_File_get_size(fout, &size) );
    MPI_Check( MPI_Barrier(comm) );

    const MPI_Offset len = content.size();
    MPI_Check( MPI_Exscan(&len, &offset, 1, MPI_OFFSET, MPI_SUM, comm) );

    MPI_Check( MPI_File_write_at_all(fout, offset + size, content.c_str(), len, MPI_CHAR, MPI_STATUS_IGNORE) );
    MPI_Check( MPI_Barrier(comm) );
}

//=================================================================================


ObjStatsDumper::ObjStatsDumper(std::string name, std::string path) :
    PostprocessPlugin(name),
    path_(makePath(path))
{}

ObjStatsDumper::~ObjStatsDumper()
{
    if (activated_)
        MPI_Check( MPI_File_close(&fout_) );
}

void ObjStatsDumper::setup(const MPI_Comm& comm, const MPI_Comm& interComm)
{
    PostprocessPlugin::setup(comm, interComm);
    activated_ = createFoldersCollective(comm, path_);
}

void ObjStatsDumper::handshake()
{
    auto req = waitData();
    MPI_Check( MPI_Wait(&req, MPI_STATUS_IGNORE) );
    recv();

    std::string ovName;
    SimpleSerializer::deserialize(data_, ovName);

    if (activated_)
    {
        const std::string fname = path_ + ovName + ".txt";
        MPI_Check( MPI_File_open(comm_, fname.c_str(), MPI_MODE_CREATE | MPI_MODE_DELETE_ON_CLOSE | MPI_MODE_WRONLY, MPI_INFO_NULL, &fout_) );
        MPI_Check( MPI_File_close(&fout_) );
        MPI_Check( MPI_File_open(comm_, fname.c_str(), MPI_MODE_WRONLY | MPI_MODE_CREATE, MPI_INFO_NULL, &fout_) );
    }
}


void ObjStatsDumper::deserialize()
{
    MirState::TimeType curTime;
    DomainInfo domain;
    std::vector<int64_t> ids;
    std::vector<COMandExtent> coms;
    std::vector<RigidMotion> motions;
    std::vector<int> typeIds;
    bool isRov;
    bool hasTypeIds;

    SimpleSerializer::deserialize(data_, curTime, domain, isRov, ids, coms, motions, hasTypeIds, typeIds);

    if (activated_)
        writeStats(comm_, domain, fout_, curTime, ids, coms, motions, isRov, hasTypeIds, typeIds);
}

} // namespace mirheo
