// Copyright 2020 ETH Zurich. All Rights Reserved.
#include "lj.h"
#include "symmetric_pairwise_helpers.h"

namespace mirheo {

PairwiseLJInteraction::PairwiseLJInteraction(const MirState *state,
                                             const std::string& name,
                                             real rc,
                                             LJParams params)
    : BasePairwiseInteraction(state, name, rc)
    , pair_(rc, params)

{}

void PairwiseLJInteraction::local(ParticleVector *pv1, ParticleVector *pv2,
                                  CellList *cl1, CellList *cl2, hipStream_t stream)
{
    pair_.setup(pv1->local(), pv2->local(), cl1, cl2, getState());
    symmetric_pairwise_helpers::computeLocalInteractions(pair_, pv1, pv2, cl1, cl2, stream);
}

void PairwiseLJInteraction::halo(ParticleVector *pv1, ParticleVector *pv2, CellList *cl1,
                                 CellList *cl2, hipStream_t stream)
{
    pair_.setup(pv1->local(), pv2->local(), cl1, cl2, getState());
    symmetric_pairwise_helpers::computeHaloInteractions(pair_, pv1, pv2, cl1, cl2, stream);
}

} // namespace mirheo
