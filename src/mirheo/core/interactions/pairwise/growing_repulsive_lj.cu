// Copyright 2020 ETH Zurich. All Rights Reserved.
#include "growing_repulsive_lj.h"
#include "symmetric_pairwise_helpers.h"

namespace mirheo {

template<class Awareness>
PairwiseGrowingRepulsiveLJInteraction<Awareness>::
PairwiseGrowingRepulsiveLJInteraction(const MirState *state,
                               const std::string& name,
                               real rc,
                               GrowingRepulsiveLJParams params,
                               std::optional<real> stressPeriod)
    : BasePairwiseInteraction(state, name, rc)
    , pair_(rc, params)

{
    if (stressPeriod)
    {
        pairWithStress_ = PairwiseStressWrapper<PairwiseGrowingRepulsiveLJ<Awareness>>(rc, params);
        stressManager_ = StressManager(*stressPeriod);
    }
}

template<class Awareness>
void PairwiseGrowingRepulsiveLJInteraction<Awareness>::
setPrerequisites(ParticleVector *pv1, ParticleVector *pv2, CellList *cl1, CellList *cl2)
{
    if (stressManager_)
    {
        pv1->requireDataPerParticle <Stress> (channel_names::stresses, DataManager::PersistenceMode::None);
        pv2->requireDataPerParticle <Stress> (channel_names::stresses, DataManager::PersistenceMode::None);

        cl1->requireExtraDataPerParticle <Stress> (channel_names::stresses);
        cl2->requireExtraDataPerParticle <Stress> (channel_names::stresses);
    }
}

template<class Awareness>
void PairwiseGrowingRepulsiveLJInteraction<Awareness>::local(ParticleVector *pv1, ParticleVector *pv2,
                                                      CellList *cl1, CellList *cl2,
                                                      hipStream_t stream)
{
    if (stressManager_)
    {
        stressManager_->computeLocalInteractions(getState(),
                                                 pair_, *pairWithStress_,
                                                 pv1, pv2, cl1, cl2, stream);
    }
    else
    {
        symmetric_pairwise_helpers::computeLocalInteractions(getState(), pair_, pv1, pv2, cl1, cl2, stream);
    }
}

template<class Awareness>
void PairwiseGrowingRepulsiveLJInteraction<Awareness>::halo(ParticleVector *pv1, ParticleVector *pv2,
                                                     CellList *cl1, CellList *cl2,
                                                     hipStream_t stream)
{
    if (stressManager_)
    {
        stressManager_->computeHaloInteractions(getState(),
                                                pair_, *pairWithStress_,
                                                pv1, pv2, cl1, cl2, stream);
    }
    else
    {
        symmetric_pairwise_helpers::computeHaloInteractions(getState(), pair_, pv1, pv2, cl1, cl2, stream);
    }
}

template<class Awareness>
std::vector<Interaction::InteractionChannel>
PairwiseGrowingRepulsiveLJInteraction<Awareness>::getOutputChannels() const
{
    std::vector<InteractionChannel> channels = {{channel_names::forces, alwaysActive}};

    if (stressManager_)
    {
        channels.push_back(stressManager_->getStressPredicate(getState()));
    }

    return channels;
}



std::unique_ptr<BasePairwiseInteraction>
makePairwiseGrowingRepulsiveLJInteraction(const MirState *state,
                                          const std::string& name,
                                          real rc, GrowingRepulsiveLJParams params)
{
    return std::visit([=](auto awarenessParams) -> std::unique_ptr<BasePairwiseInteraction>
    {
        using AwarenessParamsType = decltype(awarenessParams);
        using Awareness = typename AwarenessParamsType::KernelType;
        return std::make_unique<PairwiseGrowingRepulsiveLJInteraction<Awareness>>(state, name, rc, params);
    }, params.varAwarenessParams);
}


} // namespace mirheo
