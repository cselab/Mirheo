// Copyright 2020 ETH Zurich. All Rights Reserved.
#include "norandom_dpd.h"
#include "symmetric_pairwise_helpers.h"

namespace mirheo {

PairwiseNoRandomDPDInteraction::PairwiseNoRandomDPDInteraction(const MirState *state,
                                                               const std::string& name,
                                                               real rc,
                                                               NoRandomDPDParams params)
    : BasePairwiseInteraction(state, name, rc)
    , pair_(rc, params)

{}

void PairwiseNoRandomDPDInteraction::local(ParticleVector *pv1, ParticleVector *pv2,
                                           CellList *cl1, CellList *cl2, hipStream_t stream)
{
    pair_.setup(pv1->local(), pv2->local(), cl1, cl2, getState());
    symmetric_pairwise_helpers::computeLocalInteractions(pair_, pv1, pv2, cl1, cl2, stream);
}

void PairwiseNoRandomDPDInteraction::halo(ParticleVector *pv1, ParticleVector *pv2, CellList *cl1,
                                          CellList *cl2, hipStream_t stream)
{
    pair_.setup(pv1->local(), pv2->local(), cl1, cl2, getState());
    symmetric_pairwise_helpers::computeHaloInteractions(pair_, pv1, pv2, cl1, cl2, stream);
}

} // namespace mirheo
