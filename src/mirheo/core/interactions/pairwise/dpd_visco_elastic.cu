#include "hip/hip_runtime.h"
// Copyright 2020 ETH Zurich. All Rights Reserved.
#include "dpd_visco_elastic.h"

#include "symmetric_pairwise_helpers.h"

#include <mirheo/core/pvs/views/pv_with_pol_chain.h>
#include <mirheo/core/utils/cuda_rng.h>
#include <mirheo/core/utils/kernel_launch.h>
#include <mirheo/core/utils/cuda_common.h>

namespace mirheo {

namespace visco_elastic_dpd_kernels {

__global__ void chainFluctuationRelaxation(PVviewWithPolChainVector view, real sigma, real k, real seed)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= view.size)
        return;

    const real3 Q = view.Q[i];

    const real xix = Saru::mean0var1(seed, Q.x, Q.y);
    const real xiy = Saru::mean0var1(seed, Q.z, xix);
    const real xiz = Saru::mean0var1(seed, Q.x, xiy);

    real3 dQdt = sigma * real3{xix, xiy, xiz};
    dQdt -= k * Q;

    atomicAdd(view.dQdt + i, dQdt);
}

} // namespace visco_elastic_dpd_kernels

PairwiseViscoElasticDPDInteraction::PairwiseViscoElasticDPDInteraction(const MirState *state,
                                                                       const std::string& name,
                                                                       real rc,
                                                                       ViscoElasticDPDParams params)
    : BasePairwiseInteraction(state, name, rc)
    , params_(params)
    , pair_(rc, params)
{}

void PairwiseViscoElasticDPDInteraction::setPrerequisites(ParticleVector *pv1, ParticleVector *pv2, CellList *cl1, CellList *cl2)
{
    pv1->requireDataPerParticle <real3> (channel_names::polChainVectors, DataManager::PersistenceMode::Active);
    pv2->requireDataPerParticle <real3> (channel_names::polChainVectors, DataManager::PersistenceMode::Active);

    pv1->requireDataPerParticle <real3> (channel_names::derChainVectors, DataManager::PersistenceMode::None);
    pv2->requireDataPerParticle <real3> (channel_names::derChainVectors, DataManager::PersistenceMode::None);

    cl1->requireExtraDataPerParticle <real3> (channel_names::polChainVectors);
    cl2->requireExtraDataPerParticle <real3> (channel_names::polChainVectors);

    cl1->requireExtraDataPerParticle <real3> (channel_names::derChainVectors);
    cl2->requireExtraDataPerParticle <real3> (channel_names::derChainVectors);
}

void PairwiseViscoElasticDPDInteraction::local(ParticleVector *pv1, ParticleVector *pv2,
                                               CellList *cl1, CellList *cl2, hipStream_t stream)
{
    // single particle part; we skip the cases pv1 != pv2 to avoid accounting it multiple times.
    // We assume here that pv1 always interacts with itself exactly once with that interaction.
    if (pv1 == pv2)
    {
        const real dt = getState()->getDt();

        const real sigma = std::sqrt(4.0_r * params_.kBTC * dt / params_.zeta);
        const real k = 2.0_r * params_.H / params_.zeta;

        const auto seed = stepGen_.generate(getState());

        PVviewWithPolChainVector view(pv1, pv1->local());

        constexpr int nthreads = 128;
        const int nblocks = getNblocks(view.size, nthreads);

        SAFE_KERNEL_LAUNCH(
            visco_elastic_dpd_kernels::chainFluctuationRelaxation,
            nblocks, nthreads, 0, stream,
            view, sigma, k, seed);
    }

    symmetric_pairwise_helpers::computeLocalInteractions(getState(), pair_, pv1, pv2, cl1, cl2, stream);
}

void PairwiseViscoElasticDPDInteraction::halo(ParticleVector *pv1, ParticleVector *pv2, CellList *cl1,
                                              CellList *cl2, hipStream_t stream)
{
    symmetric_pairwise_helpers::computeHaloInteractions(getState(), pair_, pv1, pv2, cl1, cl2, stream);
}

std::vector<Interaction::InteractionChannel> PairwiseViscoElasticDPDInteraction::getInputChannels() const
{
    return {{channel_names::polChainVectors, alwaysActive}};
}

std::vector<Interaction::InteractionChannel> PairwiseViscoElasticDPDInteraction::getOutputChannels() const
{
    return {{channel_names::forces, alwaysActive},
            {channel_names::derChainVectors, alwaysActive}};
}


} // namespace mirheo
