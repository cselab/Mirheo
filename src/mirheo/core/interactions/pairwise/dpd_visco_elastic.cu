#include "hip/hip_runtime.h"
// Copyright 2020 ETH Zurich. All Rights Reserved.
#include "dpd_visco_elastic.h"

#include "symmetric_pairwise_helpers.h"

#include <mirheo/core/pvs/views/pv_with_pol_chain.h>
#include <mirheo/core/utils/cuda_rng.h>
#include <mirheo/core/utils/kernel_launch.h>
#include <mirheo/core/utils/cuda_common.h>

namespace mirheo {

namespace visco_elastic_dpd_kernels {

__global__ void chainFluctuationRelaxation(PVviewWithPolChainVector view, real sigma, real k, real seed)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= view.size)
        return;

    const real3 Q = view.Q[i];

    const real xix = Saru::mean0var1(seed, 45*i+47, 96*i-65789);
    const real xiy = Saru::mean0var1(xix, 47*i+43, 16*i-6578);
    const real xiz = Saru::mean0var1(xiy, 49*i+41, 196*i+657);

    real3 dQdt = sigma * real3{xix, xiy, xiz};
    dQdt -= k * Q;

    atomicAdd(view.dQdt + i, dQdt);
}

} // namespace visco_elastic_dpd_kernels

PairwiseViscoElasticDPDInteraction::PairwiseViscoElasticDPDInteraction(const MirState *state,
                                                                       const std::string& name,
                                                                       real rc,
                                                                       ViscoElasticDPDParams params,
                                                                       std::optional<real> stressPeriod)
    : BasePairwiseInteraction(state, name, rc)
    , params_(params)
    , pair_(rc, params)
{
    if (stressPeriod)
    {
        pairWithStress_ = PairwiseStressWrapper<PairwiseViscoElasticDPD>(rc, params);
        stressManager_ = StressManager(*stressPeriod);
    }
}

void PairwiseViscoElasticDPDInteraction::setPrerequisites(ParticleVector *pv1, ParticleVector *pv2, CellList *cl1, CellList *cl2)
{
    pv1->requireDataPerParticle <real3> (channel_names::polChainVectors, DataManager::PersistenceMode::Active);
    pv2->requireDataPerParticle <real3> (channel_names::polChainVectors, DataManager::PersistenceMode::Active);

    pv1->requireDataPerParticle <real3> (channel_names::derChainVectors, DataManager::PersistenceMode::None);
    pv2->requireDataPerParticle <real3> (channel_names::derChainVectors, DataManager::PersistenceMode::None);

    cl1->requireExtraDataPerParticle <real3> (channel_names::polChainVectors);
    cl2->requireExtraDataPerParticle <real3> (channel_names::polChainVectors);

    cl1->requireExtraDataPerParticle <real3> (channel_names::derChainVectors);
    cl2->requireExtraDataPerParticle <real3> (channel_names::derChainVectors);

    if (stressManager_)
    {
        pv1->requireDataPerParticle <Stress> (channel_names::stresses, DataManager::PersistenceMode::None);
        pv2->requireDataPerParticle <Stress> (channel_names::stresses, DataManager::PersistenceMode::None);

        cl1->requireExtraDataPerParticle <Stress> (channel_names::stresses);
        cl2->requireExtraDataPerParticle <Stress> (channel_names::stresses);
    }
}

void PairwiseViscoElasticDPDInteraction::local(ParticleVector *pv1, ParticleVector *pv2,
                                               CellList *cl1, CellList *cl2, hipStream_t stream)
{
    // single particle part; we skip the cases pv1 != pv2 to avoid accounting it multiple times.
    // We assume here that pv1 always interacts with itself exactly once with that interaction.
    if (pv1 == pv2)
    {
        const real dt = getState()->getDt();

        const real sigma = std::sqrt(4.0_r * params_.kBTC / (dt * params_.zeta));
        const real k = 2.0_r * params_.H / params_.zeta;

        const auto seed = stepGen_.generate(getState());

        PVviewWithPolChainVector view(pv1, pv1->local());

        constexpr int nthreads = 128;
        const int nblocks = getNblocks(view.size, nthreads);

        SAFE_KERNEL_LAUNCH(
            visco_elastic_dpd_kernels::chainFluctuationRelaxation,
            nblocks, nthreads, 0, stream,
            view, sigma, k, seed);
    }

    if (stressManager_)
    {
        stressManager_->computeLocalInteractions(getState(),
                                                 pair_, *pairWithStress_,
                                                 pv1, pv2, cl1, cl2, stream);
    }
    else
    {
        symmetric_pairwise_helpers::computeLocalInteractions(getState(), pair_, pv1, pv2, cl1, cl2, stream);
    }
}

void PairwiseViscoElasticDPDInteraction::halo(ParticleVector *pv1, ParticleVector *pv2, CellList *cl1,
                                              CellList *cl2, hipStream_t stream)
{
    if (stressManager_)
    {
        stressManager_->computeHaloInteractions(getState(),
                                                pair_, *pairWithStress_,
                                                pv1, pv2, cl1, cl2, stream);
    }
    else
    {
        symmetric_pairwise_helpers::computeHaloInteractions(getState(), pair_, pv1, pv2, cl1, cl2, stream);
    }
}

std::vector<Interaction::InteractionChannel> PairwiseViscoElasticDPDInteraction::getInputChannels() const
{
    return {{channel_names::polChainVectors, alwaysActive}};
}

std::vector<Interaction::InteractionChannel> PairwiseViscoElasticDPDInteraction::getOutputChannels() const
{
    std::vector<InteractionChannel> channels = {{channel_names::forces, alwaysActive},
                                                {channel_names::derChainVectors, alwaysActive}};

    if (stressManager_)
    {
        channels.push_back(stressManager_->getStressPredicate(getState()));
    }

    return channels;
}


} // namespace mirheo
