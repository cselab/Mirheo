// Copyright 2020 ETH Zurich. All Rights Reserved.
#include "factory.h"
#include "rod.h"

namespace mirheo
{

template <int Nstates>
std::shared_ptr<BaseRodInteraction>
instantiateImpl(const MirState *state, const std::string& name, RodParameters parameters,
                VarSpinParams varSpinParams, bool saveEnergies)
{
    std::shared_ptr<BaseRodInteraction> impl;

    mpark::visit([&](auto spinParams)
    {
        using SpinParamsType = decltype(spinParams);

        impl = std::make_shared<RodInteraction<Nstates, SpinParamsType>>
            (state, name, parameters, spinParams, saveEnergies);
    }, varSpinParams);

    return impl;
}


std::shared_ptr<BaseRodInteraction>
createInteractionRod(const MirState *state, const std::string& name,
                     RodParameters params, VarSpinParams spinParams, bool saveEnergies)
{
    std::shared_ptr<BaseRodInteraction> impl;
    const int nstates = params.kappaEq.size();

    if (mpark::holds_alternative<StatesParametersNone>(spinParams))
    {
        if (nstates != 1)
            die("only one state supported for state_update = 'none' (while creating %s)", name.c_str());

        impl = std::make_shared<RodInteraction<1, StatesParametersNone>>
            (state, name, params, mpark::get<StatesParametersNone>(spinParams), saveEnergies);
    }
    else
    {
        if (nstates <= 1)
            warn("using only one state for state_update != 'none' (while creating %s)", name.c_str());

#define CHECK_IMPLEMENT(Nstates) do {                                   \
            if (nstates == Nstates) {                                   \
                impl = instantiateImpl<Nstates>                         \
                    (state, name, params, spinParams, saveEnergies); \
                debug("Create interaction rod with %d states", Nstates); \
                return impl;                                                 \
            } } while(0)

        CHECK_IMPLEMENT(2); // 2 polymorphic states
        CHECK_IMPLEMENT(11); // bbacterial flagella have up to 11 states

        die("'%s' : number of states %d is not implemented", name.c_str(), nstates);
    }

    return impl;
}

} // namespace mirheo
