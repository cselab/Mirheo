// Copyright 2020 ETH Zurich. All Rights Reserved.
#include "factory.h"

#include "membrane.h"

#include "force_kernels/common.h"
#include "force_kernels/dihedral/kantor.h"
#include "force_kernels/dihedral/juelicher.h"
#include "force_kernels/triangle/lim.h"
#include "force_kernels/triangle/wlc.h"

#include <mirheo/core/utils/variant_foreach.h>

namespace mirheo
{

std::shared_ptr<BaseMembraneInteraction>
createInteractionMembrane(const MirState *state, const std::string& name,
                          CommonMembraneParameters commonParams,
                          VarBendingParams varBendingParams, VarShearParams varShearParams,
                          bool stressFree, real initLengthFraction, real growUntil, VarMembraneFilter varFilter)
{
    std::shared_ptr<BaseMembraneInteraction> impl;

    mpark::visit([&](auto bendingParams, auto shearParams, auto filter)
    {
        using DihedralForce = typename decltype(bendingParams)::DihedralForce;

        if (stressFree)
        {
            using TriangleForce = typename decltype(shearParams)::TriangleForce <StressFreeState::Active>;

            impl = std::make_shared<MembraneInteraction<TriangleForce, DihedralForce, decltype(filter)>>
                (state, name, commonParams, shearParams, bendingParams, initLengthFraction, growUntil, filter);
        }
        else
        {
            using TriangleForce = typename decltype(shearParams)::TriangleForce <StressFreeState::Inactive>;

            impl = std::make_shared<MembraneInteraction<TriangleForce, DihedralForce, decltype(filter)>>
                (state, name, commonParams, shearParams, bendingParams, initLengthFraction, growUntil, filter);
        }
    }, varBendingParams, varShearParams, varFilter);

    return std::move(impl);
}


std::shared_ptr<BaseMembraneInteraction>
loadInteractionMembrane(const MirState *state, Loader& loader, const ConfigObject& config)
{
    std::shared_ptr<BaseMembraneInteraction> impl;
    const std::string& typeName = config["__type"].getString();

    // The arguments are type_identity<T> for three different types T, empty
    // structs carrying the type information.
    auto visitor = [&](auto bending, auto shear, auto filter)
    {
        using DihedralForce = typename decltype(bending)::type::DihedralForce;
        using ShearParams   = typename decltype(shear)::type;
        using FilterType  = typename decltype(filter)::type;

        {
            using TriangleForce = typename ShearParams::TriangleForce <StressFreeState::Active>;
            using Impl = MembraneInteraction<TriangleForce, DihedralForce, FilterType>;
            if (Impl::getTypeName() == typeName)
            {
                impl = std::make_shared<Impl>(state, loader, config);
                return;
            }
        }
        {
            using TriangleForce = typename ShearParams::TriangleForce <StressFreeState::Inactive>;
            using Impl = MembraneInteraction<TriangleForce, DihedralForce, FilterType>;
            if (Impl::getTypeName() == typeName)
            {
                impl = std::make_shared<Impl>(state, loader, config);
                return;
            }
        }
    };

    // Check all possible template combinations and match with the `typeName`.
    variantForeach<VarBendingParams, VarShearParams, VarMembraneFilter>(visitor);

    if (!impl)
        die("Unrecognized impl type \"%s\".", typeName.c_str());

    return std::move(impl);
}

} // namespace mirheo
