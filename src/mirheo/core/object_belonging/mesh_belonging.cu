#include "hip/hip_runtime.h"
// Copyright 2020 ETH Zurich. All Rights Reserved.
#include "mesh_belonging.h"

#include <mirheo/core/celllist.h>
#include <mirheo/core/pvs/object_vector.h>
#include <mirheo/core/pvs/views/ov.h>
#include <mirheo/core/rigid/utils.h>
#include <mirheo/core/utils/kernel_launch.h>
#include <mirheo/core/utils/quaternion.h>

namespace mirheo
{

namespace mesh_belonging_kernels
{

const real tolerance = 1e-6_r;

/// https://en.wikipedia.org/wiki/M%C3%B6ller%E2%80%93Trumbore_intersection_algorithm
__device__ static inline bool doesRayIntersectTriangle(
        real3 rayOrigin,
        real3 rayVector,
        real3 v0, real3 v1, real3 v2)
{
    real3 edge1, edge2, h, s, q;
    real a,f,u,v;

    edge1 = v1 - v0;
    edge2 = v2 - v0;
    h = cross(rayVector, edge2);
    a = dot(edge1, h);
    if (math::abs(a) < tolerance)
        return false;

    f = 1.0_r / a;
    s = rayOrigin - v0;
    u = f * (dot(s, h));
    if (u < 0.0_r || u > 1.0_r)
        return false;

    q = cross(s, edge1);
    v = f * dot(rayVector, q);
    if (v < 0.0_r || u + v > 1.0_r)
        return false;

    // At this stage we can compute t to find out where the intersection point is on the line.
    real t = f * dot(edge2, q);

    if (t > tolerance) // ray intersection
        return true;
    else
        return false; // This means that there is a line intersection but not a ray intersection.
}


__device__ static inline real3 fetchPosition(const real4 *vertices, int i)
{
    auto v = vertices[i];
    return {v.x, v.y, v.z};
}

/**
 * One warp works on one particle
 */
__device__ static inline BelongingTags oneParticleInsideMesh(int pid, real3 r, int objId, const real3 com, const MeshView mesh, const real4* vertices)
{
    // Work in obj reference frame for simplicity
    r = r - com;

    // shoot 3 rays in different directions, count intersections
    constexpr int nRays = 3;
    constexpr real3 rays[nRays] = { {0,1,0}, {0,1,0}, {0,1,0} };
    int counters[nRays] = {0, 0, 0};

    for (int i = laneId(); i < mesh.ntriangles; i += warpSize)
    {
        int3 trid = mesh.triangles[i];

        real3 v0 = fetchPosition(vertices, objId*mesh.nvertices + trid.x) - com;
        real3 v1 = fetchPosition(vertices, objId*mesh.nvertices + trid.y) - com;
        real3 v2 = fetchPosition(vertices, objId*mesh.nvertices + trid.z) - com;

        for (int c = 0; c < nRays; c++)
            if (doesRayIntersectTriangle(r, rays[c], v0, v1, v2))
                counters[c]++;
    }

    // counter is odd if the particle is inside
    // however, realing-point precision sometimes yields in errors
    // so we choose what the majority(!) of the rays say
    int intersecting = 0;
    for (int c = 0; c < nRays; c++)
    {
        counters[c] = warpReduce(counters[c], [] (int a, int b) { return a+b; });
        if ( (counters[c] % 2) != 0 )
            intersecting++;
    }

    if (intersecting > (nRays/2))
        return BelongingTags::Inside;
    else
        return BelongingTags::Outside;
}

/**
 * OVview view is only used to provide # of objects and extent information
 * Actual data is in \p vertices
 * @param cinfo is the cell-list sync'd with the target ParticleVector data
 */
template<int WARPS_PER_OBJ>
__global__ void insideMesh(const OVview ovView, const MeshView mesh, const real4 *vertices, CellListInfo cinfo, PVview pvView, BelongingTags* tags)
{
    const int gid = blockIdx.x*blockDim.x + threadIdx.x;
    const int wid = gid / warpSize;
    const int objId = wid / WARPS_PER_OBJ;

    const int locWid = wid % WARPS_PER_OBJ;

    if (objId >= ovView.nObjects) return;

    const int3 cidLow  = cinfo.getCellIdAlongAxes(ovView.comAndExtents[objId].low  - 0.5_r);
    const int3 cidHigh = cinfo.getCellIdAlongAxes(ovView.comAndExtents[objId].high + 0.5_r);

    const int3 span = cidHigh - cidLow + make_int3(1,1,1);
    const int totCells = span.x * span.y * span.z;

    for (int i = locWid; i < totCells; i += WARPS_PER_OBJ)
    {
        const int3 cid3 = make_int3( i % span.x, (i/span.x) % span.y, i / (span.x*span.y) ) + cidLow;
        const int  cid = cinfo.encode(cid3);
        if (cid < 0 || cid >= cinfo.totcells) continue;

        int pstart = cinfo.cellStarts[cid];
        int pend   = cinfo.cellStarts[cid+1];

#pragma unroll 3
        for (int pid = pstart; pid < pend; pid++)
        {
            const Particle p(pvView.readParticle(pid));

            auto tag = oneParticleInsideMesh(pid, p.r, objId, ovView.comAndExtents[objId].com, mesh, vertices);

            // Only tag particles inside, default is outside anyways
            if (laneId() == 0 && tag != BelongingTags::Outside)
                tags[pid] = tag;
        }
    }
}

} // namespace mesh_belonging_kernels

void MeshBelongingChecker::_tagInner(ParticleVector *pv, CellList *cl, hipStream_t stream)
{
    tags_.resize_anew(pv->local()->size());
    tags_.clearDevice(stream);

    auto computeTags = [&](ParticleVectorLocality locality)
    {
        ov_->findExtentAndCOM(stream, locality);

        auto lov = ov_->get(locality);
        auto view = OVview(ov_, lov);
        auto vertices = lov->getMeshVertices(stream);
        auto meshView = MeshView(ov_->mesh.get());

        debug("Computing inside/outside tags (against mesh) for %d %s objects '%s' and %d '%s' particles",
              view.nObjects, getParticleVectorLocalityStr(locality).c_str(),
              ov_->getCName(), pv->local()->size(), pv->getCName());

        constexpr int nthreads = 128;
        constexpr int warpsPerObject = 1024;

        SAFE_KERNEL_LAUNCH(
            mesh_belonging_kernels::insideMesh<warpsPerObject>,
            getNblocks(warpsPerObject*32*view.nObjects, nthreads), nthreads, 0, stream,
            view, meshView, reinterpret_cast<real4*>(vertices->devPtr()),
            cl->cellInfo(), cl->getView<PVview>(), tags_.devPtr());
    };


    computeTags(ParticleVectorLocality::Local);
    computeTags(ParticleVectorLocality::Halo);
}

} // namespace mirheo
