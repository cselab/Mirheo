#include "hip/hip_runtime.h"
// Copyright 2020 ETH Zurich. All Rights Reserved.
#include "celllist.h"

#include <mirheo/core/logger.h>
#include <mirheo/core/pvs/object_vector.h>
#include <mirheo/core/pvs/particle_vector.h>
#include <mirheo/core/pvs/views/pv.h>
#include <mirheo/core/utils/cuda_common.h>
#include <mirheo/core/utils/kernel_launch.h>
#include <mirheo/core/utils/type_traits.h>

#pragma GCC diagnostic push
#pragma GCC diagnostic ignored "-Wshadow"
#include <cub/device/device_scan.cuh>
#pragma GCC diagnostic pop

#include <algorithm>

namespace mirheo
{

namespace cell_list_kernels
{

enum {INVALID = -1};

inline __device__ bool outgoingParticle(real4 pos)
{
    return Real3_int(pos).isMarked();
}

__global__ void computeCellSizes(PVview view, CellListInfo cinfo)
{
    const int pid = blockIdx.x * blockDim.x + threadIdx.x;
    if (pid >= view.size) return;

    real4 coo = view.readPositionNoCache(pid);

    // XXX: relying here only on redistribution
    if ( outgoingParticle(coo) ) return;

    int cid = cinfo.getCellId<CellListsProjection::Clamp>(coo);
    atomicAdd(cinfo.cellSizes + cid, 1);
}

__global__ void reorderPositionsAndCreateMap(PVview view, CellListInfo cinfo, real4 *outPositions)
{
    const int pid = blockIdx.x * blockDim.x + threadIdx.x;
    if (pid >= view.size) return;

    int dstId = INVALID;

    // this is to allow more cache for atomics
    // loads / stores here need no cache
    real4 pos = view.readPositionNoCache(pid);

    int cid = cinfo.getCellId<CellListsProjection::Clamp>(pos);

    //  XXX: relying here only on redistribution
    if ( !outgoingParticle(pos) )
        dstId = cinfo.cellStarts[cid] + atomicAdd(cinfo.cellSizes + cid, 1);

    if (dstId != INVALID)
        writeNoCache(outPositions + dstId, pos);

    cinfo.order[pid] = dstId;
}

template <typename T>
__global__ void reorderExtraDataPerParticle(int n, const T *inExtraData, CellListInfo cinfo, T *outExtraData)
{
    int srcId = blockIdx.x * blockDim.x + threadIdx.x;
    if (srcId >= n) return;

    int dstId = cinfo.order[srcId];
    if (dstId != INVALID)
        outExtraData[dstId] = inExtraData[srcId];
}

template <typename T>
__global__ void accumulateKernel(int n, T *dst, CellListInfo cinfo, const T *src)
{
    int pid = blockIdx.x * blockDim.x + threadIdx.x;
    if (pid >= n) return;

    int srcId = cinfo.order[pid];

    assert(srcId != INVALID);
    dst[pid] += src[srcId];
}

} // namespace cell_list_kernels

//=================================================================================
// Info
//=================================================================================

CellListInfo::CellListInfo(real rc_, real3 localDomainSize_) :
    rc(rc_),
    localDomainSize(localDomainSize_)
{
    ncells = make_int3( math::floor(localDomainSize / rc + 1e-6_r) );
    h = make_real3(localDomainSize) / make_real3(ncells);
    invh_ = 1.0_r / h;
    rc = std::min( {h.x, h.y, h.z} );

    totcells = ncells.x * ncells.y * ncells.z;
}

CellListInfo::CellListInfo(real3 h_, real3 localDomainSize_) :
    h(h_),
    invh_(1.0_r/h_),
    localDomainSize(localDomainSize_)
{
    ncells = make_int3( math::ceil(localDomainSize / h - 1e-6_r) );
    totcells = ncells.x * ncells.y * ncells.z;
    h = make_real3(localDomainSize) / make_real3(ncells); // in case h does not divide localDomainSize
    invh_ = 1.0_r / h;
    rc = std::min( {h.x, h.y, h.z} );
}

//=================================================================================
// Basic cell-lists
//=================================================================================

CellList::CellList(ParticleVector *pv, real rc_, real3 localDomainSize_) :
    CellListInfo(rc_, localDomainSize_),
    pv_(pv),
    particlesDataContainer_(std::make_unique<LocalParticleVector>(nullptr))
{
    _initialize();
}

CellList::CellList(ParticleVector *pv, int3 resolution, real3 localDomainSize_) :
    CellListInfo(localDomainSize_ / make_real3(resolution), localDomainSize_),
    pv_(pv),
    particlesDataContainer_(std::make_unique<LocalParticleVector>(nullptr))
{
    _initialize();
}

void CellList::_initialize()
{
    localPV_ = particlesDataContainer_.get();

    cellSizes. resize_anew(totcells + 1);
    cellStarts.resize_anew(totcells + 1);

    cellSizes. clear(defaultStream);
    cellStarts.clear(defaultStream);
    CUDA_Check( hipStreamSynchronize(defaultStream) );

    debug("Initialized %s cell-list with %dx%dx%d cells and cut-off %f", pv_->getCName(), ncells.x, ncells.y, ncells.z, rc);
}

CellList::~CellList() = default;

bool CellList::_checkNeedBuild() const
{
    if (changedStamp_ == pv_->cellListStamp)
    {
        debug2("%s is already up-to-date, building skipped", _makeName().c_str());
        return false;
    }

    if (pv_->local()->size() == 0)
    {
        debug2("%s consists of no particles, building skipped", _makeName().c_str());
        return false;
    }

    return true;
}

void CellList::_updateExtraDataChannels(__UNUSED hipStream_t stream)
{
    auto& pvManager        = pv_->local()->dataPerParticle;
    auto& containerManager = particlesDataContainer_->dataPerParticle;
    const int np = pv_->local()->size();

    for (const auto& namedChannel : pvManager.getSortedChannels())
    {
        const auto& name = namedChannel.first;
        const auto& desc = namedChannel.second;
        if (desc->persistence != DataManager::PersistenceMode::Active) continue;

        mpark::visit([&](auto pinnedBuffPtr)
        {
            using T = typename std::remove_pointer<decltype(pinnedBuffPtr)>::type::value_type;

            if (!containerManager.checkChannelExists(name))
                containerManager.createData<T>(name, np);

        }, desc->varDataPtr);
    }
}

void CellList::_computeCellSizes(hipStream_t stream)
{
    debug2("%s : Computing cell sizes for %d particles", _makeName().c_str(), pv_->local()->size());
    cellSizes.clear(stream);

    PVview view(pv_, pv_->local());

    const int nthreads = 128;
    SAFE_KERNEL_LAUNCH(
            cell_list_kernels::computeCellSizes,
            getNblocks(view.size, nthreads), nthreads, 0, stream,
            view, cellInfo() );
}

void CellList::_computeCellStarts(hipStream_t stream)
{
    // Scan is always working with the same number of cells
    // Memory requirements can't change
    size_t bufSize = scanBuffer.size();

    if (bufSize == 0)
    {
        hipcub::DeviceScan::ExclusiveSum(nullptr, bufSize, cellSizes.devPtr(), cellStarts.devPtr(), totcells+1, stream);
        scanBuffer.resize_anew(bufSize);
    }
    hipcub::DeviceScan::ExclusiveSum(scanBuffer.devPtr(), bufSize,
                                  cellSizes.devPtr(), cellStarts.devPtr(), totcells+1, stream);
}

void CellList::_reorderPositionsAndCreateMap(hipStream_t stream)
{
    debug2("Reordering %d %s particles", pv_->local()->size(), pv_->getCName());

    PVview view(pv_, pv_->local());

    order.resize_anew(view.size);
    particlesDataContainer_->resize_anew(view.size);
    cellSizes.clear(stream);

    const int nthreads = 128;
    SAFE_KERNEL_LAUNCH(
        cell_list_kernels::reorderPositionsAndCreateMap,
        getNblocks(view.size, nthreads), nthreads, 0, stream,
        view, cellInfo(), particlesDataContainer_->positions().devPtr() );
}

void CellList::_reorderExtraDataEntry(const std::string& channelName,
                                      const DataManager::ChannelDescription *channelDesc,
                                      hipStream_t stream)
{
    const auto& dstDesc = particlesDataContainer_->dataPerParticle.getChannelDescOrDie(channelName);
    const int np = pv_->local()->size();

    debug2("%s: reordering extra data '%s'", _makeName().c_str(), channelName.c_str());

    mpark::visit([&](auto srcPinnedBuff)
    {
        auto dstPinnedBuff = mpark::get<decltype(srcPinnedBuff)>(dstDesc.varDataPtr);

        constexpr int nthreads = 128;

        SAFE_KERNEL_LAUNCH(
           cell_list_kernels::reorderExtraDataPerParticle,
           getNblocks(np, nthreads), nthreads, 0, stream,
           np, srcPinnedBuff->devPtr(), this->cellInfo(), dstPinnedBuff->devPtr() );
    }, channelDesc->varDataPtr);
}

void CellList::_reorderPersistentData(hipStream_t stream)
{
    auto srcExtraData = &pv_->local()->dataPerParticle;

    for (const auto& namedChannel : srcExtraData->getSortedChannels())
    {
        const auto& name = namedChannel.first;
        const auto& desc = namedChannel.second;
        if (desc->persistence != DataManager::PersistenceMode::Active
            || name == channel_names::positions) // positions were already reordered manually
            continue;
        _reorderExtraDataEntry(name, desc, stream);
    }
}

void CellList::_build(hipStream_t stream)
{
    _computeCellSizes(stream);
    _computeCellStarts(stream);
    _reorderPositionsAndCreateMap(stream);
    _reorderPersistentData(stream);

    changedStamp_ = pv_->cellListStamp;
}

CellListInfo CellList::cellInfo()
{
    CellListInfo::cellSizes  = cellSizes.devPtr();
    CellListInfo::cellStarts = cellStarts.devPtr();
    CellListInfo::order      = order.devPtr();

    return *((CellListInfo*)this);
}

void CellList::build(hipStream_t stream)
{
    _updateExtraDataChannels(stream);

    if (!_checkNeedBuild()) return;

    debug("building %s", _makeName().c_str());

    _build(stream);
}

static void accumulateIfHasAddOperator(__UNUSED GPUcontainer *src,
                                       __UNUSED GPUcontainer *dst,
                                       __UNUSED int n, __UNUSED CellListInfo cinfo,
                                       __UNUSED hipStream_t stream)
{
    die("Cannot accumulate entries: operator+ not supported for this type");
}

// use SFINAE to choose between additionable types
template <typename T, typename = void_t<decltype(std::declval<T>() +
                                                 std::declval<T>())>>
static void accumulateIfHasAddOperator(PinnedBuffer<T> *src,
                                       PinnedBuffer<T> *dst,
                                       int n, CellListInfo cinfo,
                                       hipStream_t stream)
{
    const int nthreads = 128;

    SAFE_KERNEL_LAUNCH(
        cell_list_kernels::accumulateKernel,
        getNblocks(n, nthreads), nthreads, 0, stream,
        n, dst->devPtr(), cinfo, src->devPtr() );
}

void CellList::_accumulateExtraData(const std::string& channelName, hipStream_t stream)
{
    const int n = pv_->local()->size();

    const auto& pvManager   = pv_->local()->dataPerParticle;
    const auto& contManager = localPV_->dataPerParticle;

    const auto& pvDesc   = pvManager  .getChannelDescOrDie(channelName);
    const auto& contDesc = contManager.getChannelDescOrDie(channelName);

    mpark::visit([&](auto srcPinnedBuff)
    {
        auto dstPinnedBuff = mpark::get<decltype(srcPinnedBuff)>(pvDesc.varDataPtr);
        accumulateIfHasAddOperator(srcPinnedBuff, dstPinnedBuff, n, this->cellInfo(), stream);
    }, contDesc.varDataPtr);
}

void CellList::accumulateChannels(const std::vector<std::string>& channelNames, hipStream_t stream)
{
    for (const auto& channelName : channelNames)
    {
        debug2("%s : accumulating channel '%s'", _makeName().c_str(), channelName.c_str());
        _accumulateExtraData(channelName, stream);
    }
}

void CellList::gatherChannels(const std::vector<std::string>& channelNames, hipStream_t stream)
{
    for (auto& channelName : channelNames)
    {
        debug("%s : gathering channel '%s'", _makeName().c_str(), channelName.c_str());

        auto& desc = localPV_->dataPerParticle.getChannelDescOrDie(channelName);
        _reorderExtraDataEntry(channelName, &desc, stream);

        // invalidate particle vector halo if any entry is active
        pv_->haloValid = false;
    }
}

void CellList::clearChannels(const std::vector<std::string>& channelNames, hipStream_t stream)
{
    for (const auto& channelName : channelNames)
    {
        debug2("%s : clearing channel '%s'", _makeName().c_str(), channelName.c_str());
        localPV_->dataPerParticle.getGenericData(channelName)->clearDevice(stream);
    }
}

LocalParticleVector* CellList::getLocalParticleVector() {return localPV_;}

std::string CellList::_makeName() const
{
    return "Cell List '" + pv_->getName() + "' (rc " + std::to_string(rc) + ")";
}


//=================================================================================
// Primary cell-lists
//=================================================================================

PrimaryCellList::PrimaryCellList(ParticleVector *pv, real rc_, real3 localDomainSize_) :
        CellList(pv, rc_, localDomainSize_)
{
    localPV_ = pv_->local();

    if (dynamic_cast<ObjectVector*>(pv_) != nullptr)
        error("Using primary cell-lists with objects is STRONGLY discouraged. This will very likely result in an error");
}

PrimaryCellList::PrimaryCellList(ParticleVector *pv, int3 resolution, real3 localDomainSize_) :
        CellList(pv, resolution, localDomainSize_)
{
    localPV_ = pv_->local();

    if (dynamic_cast<ObjectVector*>(pv_) != nullptr)
        error("Using primary cell-lists with objects is STRONGLY discouraged. This will very likely result in an error");
}

PrimaryCellList::~PrimaryCellList() = default;

void PrimaryCellList::build(hipStream_t stream)
{
    // Reqired here to avoid ptr swap if building didn't actually happen
    if (!_checkNeedBuild()) return;

    CellList::build(stream);

    if (pv_->local()->size() == 0)
    {
        debug2("%s consists of no particles, cell-list building skipped", pv_->getCName());
        return;
    }

    // Now we need the new size of particles array.
    int newSize;
    CUDA_Check( hipMemcpyAsync(&newSize, cellStarts.devPtr() + totcells, sizeof(int), hipMemcpyDeviceToHost, stream) );
    CUDA_Check( hipStreamSynchronize(stream) );

    debug2("%s : reordering completed, new size of %s particle vector is %d",
           _makeName().c_str(), pv_->getCName(), newSize);

    particlesDataContainer_->resize(newSize, stream);

    _swapPersistentExtraData();

    pv_->local()->resize(newSize, stream);
}

void PrimaryCellList::accumulateChannels(__UNUSED const std::vector<std::string>& channelNames, __UNUSED hipStream_t stream)
{}

void PrimaryCellList::gatherChannels(const std::vector<std::string>& channelNames, __UNUSED hipStream_t stream)
{
    // do not need to reorder data, but still invalidate halo
    if (!channelNames.empty())
        pv_->haloValid = false;
}


template <typename T>
static void swap(const std::string& channelName, DataManager& pvManager, DataManager& containerManager)
{
    std::swap(*pvManager       .getData<T>(channelName),
              *containerManager.getData<T>(channelName));
}

void PrimaryCellList::_swapPersistentExtraData()
{
    auto& pvManager        = pv_->local()->dataPerParticle;
    auto& containerManager = particlesDataContainer_->dataPerParticle;

    for (const auto& namedChannel : pvManager.getSortedChannels())
    {
        const auto& name = namedChannel.first;
        const auto& desc = namedChannel.second;
        if (desc->persistence != DataManager::PersistenceMode::Active)
            continue;

        const auto& descCont = containerManager.getChannelDescOrDie(name);

        mpark::visit([&](auto pinnedBufferPv)
        {
            auto pinnedBufferCont = mpark::get<decltype(pinnedBufferPv)>(descCont.varDataPtr);
            std::swap(*pinnedBufferPv, *pinnedBufferCont);
        }, desc->varDataPtr);
    }
}

std::string PrimaryCellList::_makeName() const
{
    return "Primary " + CellList::_makeName();
}

} // namespace mirheo
