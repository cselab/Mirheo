#include "hip/hip_runtime.h"
// Copyright 2020 ETH Zurich. All Rights Reserved.
#include "particle_redistributor.h"

#include "exchange_entity.h"
#include "utils/common.h"
#include "utils/face_dispatch.h"
#include "utils/fragments_mapping.h"

#include <mirheo/core/celllist.h>
#include <mirheo/core/pvs/packers/particles.h>
#include <mirheo/core/pvs/particle_vector.h>
#include <mirheo/core/pvs/views/pv.h>
#include <mirheo/core/utils/cuda_common.h>
#include <mirheo/core/utils/kernel_launch.h>

#include <algorithm>

namespace mirheo
{

enum class PackMode
{
    Query, Pack
};

namespace particle_redistributor_kernels
{
inline __device__ int encodeCellId1d(int cid, int ncells)
{
    if      (cid <  0     ) return -1;
    else if (cid >= ncells) return  1;
    else                    return  0;
}

inline __device__ int3 encodeCellId(int3 cid, int3 ncells)
{
    cid.x = encodeCellId1d(cid.x, ncells.x);
    cid.y = encodeCellId1d(cid.y, ncells.y);
    cid.z = encodeCellId1d(cid.z, ncells.z);
    return cid;
}

inline __device__ bool hasToLeave(int3 dir)
{
    return dir.x != 0 || dir.y != 0 || dir.z != 0;
}

template <PackMode packMode>
__global__ void getExitingParticles(CellListInfo cinfo, PVview view, DomainInfo domain,
                                    ParticlePackerHandler packer, BufferOffsetsSizesWrap dataWrap)
{
    const int gid = blockIdx.x*blockDim.x + threadIdx.x;
    const int faceId = blockIdx.y;
    int cid;
    int dx, dy, dz;

    bool valid = distributeThreadsToFaceCell(cid, dx, dy, dz, gid, faceId, cinfo);

    if (!valid) return;

    // The following is called for every outer cell and exactly once for each
    // Now for each cell we check its every particle if it needs to move

    int pstart = cinfo.cellStarts[cid];
    int pend   = cinfo.cellStarts[cid+1];

#pragma unroll 2
    for (int i = 0; i < pend-pstart; i++)
    {
        const int srcId = pstart + i;
        Particle p;
        view.readPosition(p, srcId);

        int3 dir = cinfo.getCellIdAlongAxes<CellListsProjection::NoClamp>(p.r);

        dir = encodeCellId(dir, cinfo.ncells);

        if (p.isMarked()) continue;

        if (hasToLeave(dir))
        {
            const int bufId = fragment_mapping::getId(dir);

            int myId = atomicAdd(dataWrap.sizes + bufId, 1);

            if (packMode == PackMode::Query)
            {
                continue;
            }
            else
            {
                auto shift = exchangers_common::getShift(domain.localSize, dir);

                const int numElements = dataWrap.offsets[bufId+1] - dataWrap.offsets[bufId];

                auto buffer = dataWrap.getBuffer(bufId);

                packer.particles.packShift(srcId, myId, buffer, numElements, shift);

                // mark the particle as exited to assist cell-list building
                Real3_int pos = p.r2Real3_int();
                pos.mark();
                view.writePosition(srcId, pos.toReal4());
            }
        }
    }
}

__global__ void unpackParticles(int startDstId, BufferOffsetsSizesWrap dataWrap,
                                ParticlePackerHandler packer)
{
    const int bufId = blockIdx.x;

    const int numElements = dataWrap.sizes[bufId];

    for (int pid = threadIdx.x; pid < numElements; pid += blockDim.x)
    {
        const int dstId = startDstId + dataWrap.offsets[bufId] + pid;
        const auto buffer = dataWrap.getBuffer(bufId);

        packer.particles.unpack(pid, dstId, buffer, numElements);
    }
}

} // namespace particle_redistributor_kernels

//===============================================================================================
// Member functions
//===============================================================================================

ParticleRedistributor::ParticleRedistributor() = default;
ParticleRedistributor::~ParticleRedistributor() = default;

bool ParticleRedistributor::needExchange(size_t id)
{
    return !particles_[id]->redistValid;
}

void ParticleRedistributor::attach(ParticleVector *pv, CellList *cl)
{
    const size_t id = particles_.size();
    particles_.push_back(pv);
    cellLists_.push_back(cl);

    if (dynamic_cast<PrimaryCellList*>(cl) == nullptr)
        die("Redistributor (for %s) must be used with a primary cell-list", pv->getCName());

    PackPredicate predicate = [](const DataManager::NamedChannelDesc& namedDesc)
    {
        return namedDesc.second->persistence == DataManager::PersistenceMode::Active;
    };

    auto packer = std::make_unique<ParticlePacker>(predicate);
    auto helper = std::make_unique<ExchangeEntity>(pv->getName(), id, packer.get());

    packers_.push_back(std::move(packer));
    this->addExchangeEntity(std::move(helper));

    info("Particle redistributor takes pv '%s'", pv->getCName());
}

void ParticleRedistributor::prepareSizes(size_t id, hipStream_t stream)
{
    auto pv = particles_[id];
    auto cl = cellLists_[id];
    auto helper = getExchangeEntity(id);
    auto packer = packers_[id].get();
    auto lpv = pv->local();

    debug2("Counting leaving particles of '%s'", pv->getCName());

    helper->send.sizes.clear(stream);

    packer->update(lpv, stream);

    if (lpv->size() > 0)
    {
        const int maxdim = std::max({cl->ncells.x, cl->ncells.y, cl->ncells.z});
        const int nthreads = 64;
        const dim3 nblocks = dim3(getNblocks(maxdim*maxdim, nthreads), 6, 1);

        SAFE_KERNEL_LAUNCH(
            particle_redistributor_kernels::getExitingParticles<PackMode::Query>,
            nblocks, nthreads, 0, stream,
            cl->cellInfo(), cl->getView<PVview>(),
            pv->getState()->domain, packer->handler(),
            helper->wrapSendData() );
    }
    helper->computeSendOffsets_Dev2Dev(stream);
}

void ParticleRedistributor::prepareData(size_t id, hipStream_t stream)
{
    auto pv = particles_[id];
    auto cl = cellLists_[id];
    auto helper = getExchangeEntity(id);
    auto packer = packers_[id].get();

    debug2("Downloading %d leaving particles of '%s'",
           helper->send.offsets[helper->nBuffers], pv->getCName());

    if (pv->local()->size() > 0)
    {
        const int maxdim = std::max({cl->ncells.x, cl->ncells.y, cl->ncells.z});
        const int nthreads = 64;
        const dim3 nblocks = dim3(getNblocks(maxdim*maxdim, nthreads), 6, 1);

        helper->resizeSendBuf();

        // Sizes will still remain on host, no need to download again
        helper->send.sizes.clearDevice(stream);

        SAFE_KERNEL_LAUNCH(
            particle_redistributor_kernels::getExitingParticles<PackMode::Pack>,
            nblocks, nthreads, 0, stream,
            cl->cellInfo(), cl->getView<PVview>(),
            pv->getState()->domain, packer->handler(),
            helper->wrapSendData() );
    }
}

void ParticleRedistributor::combineAndUploadData(size_t id, hipStream_t stream)
{
    auto pv = particles_[id];
    auto helper = getExchangeEntity(id);
    auto packer = packers_[id].get();
    auto lpv = pv->local();

    int oldSize = lpv->size();
    int totalRecvd = helper->recv.offsets[helper->nBuffers];
    lpv->resize(oldSize + totalRecvd, stream);

    if (totalRecvd > 0)
    {
        const int nthreads = 64;
        const int nblocks  = helper->nBuffers - 1;

        packer->update(lpv, stream);

        SAFE_KERNEL_LAUNCH(
            particle_redistributor_kernels::unpackParticles,
            nblocks, nthreads, 0, stream,
            oldSize, helper->wrapRecvData(), packer->handler());

        // Particles may have migrated, rebuild cell-lists
        pv->cellListStamp++;
    }

    pv->redistValid = true;
}

} // namespace mirheo
