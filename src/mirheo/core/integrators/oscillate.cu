#include "hip/hip_runtime.h"
#include "oscillate.h"
#include "integration_kernel.h"

#include <mirheo/core/logger.h>
#include <mirheo/core/pvs/particle_vector.h>

namespace mirheo
{

IntegratorOscillate::IntegratorOscillate(const MirState *state, const std::string& name, real3 vel, real period) :
    Integrator(state, name),
    vel_(vel),
    period_(period)
{
    if (period_ <= 0)
        die("Oscillating period should be strictly positive");
}

IntegratorOscillate::~IntegratorOscillate() = default;

/**
 * Oscillate with cos wave in time, regardless force
 */
void IntegratorOscillate::execute(ParticleVector *pv, hipStream_t stream)
{
    const auto t = static_cast<real>(getState()->currentTime);

    const auto vel = vel_;
    constexpr auto twoPi = static_cast<real>(2.0 * M_PI);

    const real cosOmega = math::cos(twoPi * t / period_);

    auto oscillate = [vel, cosOmega] __device__ (Particle& p, real3 f, real invm, real dt)
    {
        p.u = vel * cosOmega;
        p.r += p.u * dt;
    };

    integrate(pv, getState()->dt, oscillate, stream);
    invalidatePV_(pv);
}

} // namespace mirheo
