#include "hip/hip_runtime.h"
// Copyright 2020 ETH Zurich. All Rights Reserved.
#include "wall_with_velocity.h"

#include "common_kernels.h"
#include "stationary_walls/box.h"
#include "stationary_walls/cylinder.h"
#include "stationary_walls/plane.h"
#include "stationary_walls/sdf.h"
#include "stationary_walls/sphere.h"
#include "velocity_field/oscillate.h"
#include "velocity_field/rotate.h"
#include "velocity_field/translate.h"

#include <mirheo/core/celllist.h>
#include <mirheo/core/logger.h>
#include <mirheo/core/pvs/object_vector.h>
#include <mirheo/core/pvs/particle_vector.h>
#include <mirheo/core/pvs/views/pv.h>
#include <mirheo/core/utils/cuda_common.h>
#include <mirheo/core/utils/kernel_launch.h>
#include <mirheo/core/utils/root_finder.h>

#include <cassert>
#include <cmath>
#include <fstream>
#include <texture_types.h>

namespace mirheo
{


template<typename VelocityField>
__global__ void imposeVelField(PVview view, const VelocityField velField)
{
    const int pid = blockIdx.x * blockDim.x + threadIdx.x;
    if (pid >= view.size) return;

    Particle p(view.readParticle(pid));

    p.u = velField(p.r);

    view.writeParticle(pid, p);
}

//===============================================================================================
// Member functions
//===============================================================================================

template<class InsideWallChecker, class VelocityField>
WallWithVelocity<InsideWallChecker, VelocityField>::WallWithVelocity
(const MirState *state, const std::string& name, InsideWallChecker&& insideWallChecker, VelocityField&& velField) :
    SimpleStationaryWall<InsideWallChecker>(state, name, std::move(insideWallChecker)),
    velField_(std::move(velField))
{}


template<class InsideWallChecker, class VelocityField>
void WallWithVelocity<InsideWallChecker, VelocityField>::setup(MPI_Comm& comm)
{
    info("Setting up wall %s", this->getCName());

    CUDA_Check( hipDeviceSynchronize() );

    this->insideWallChecker_.setup(comm, this->getState()->domain);
    velField_.setup(this->getState()->currentTime, this->getState()->domain);

    CUDA_Check( hipDeviceSynchronize() );
}

template<class InsideWallChecker, class VelocityField>
void WallWithVelocity<InsideWallChecker, VelocityField>::attachFrozen(ParticleVector *pv)
{
    SimpleStationaryWall<InsideWallChecker>::attachFrozen(pv);

    const int nthreads = 128;
    PVview view(pv, pv->local());
    SAFE_KERNEL_LAUNCH(
            imposeVelField,
            getNblocks(view.size, nthreads), nthreads, 0, 0,
            view, velField_.handler() );

    CUDA_Check( hipDeviceSynchronize() );
}

template<class InsideWallChecker, class VelocityField>
void WallWithVelocity<InsideWallChecker, VelocityField>::bounce(hipStream_t stream)
{
    real t  = this->getState()->currentTime;
    real dt = this->getState()->getDt();

    velField_.setup(t, this->getState()->domain);
    this->bounceForce_.clear(stream);

    for (size_t i = 0; i < this->particleVectors_.size(); ++i)
    {
        auto  pv = this->particleVectors_[i];
        auto  cl = this->cellLists_[i];
        auto& bc = this->boundaryCells_[i];
        auto view = cl->CellList::getView<PVviewWithOldParticles>();

        debug2("Bouncing %d %s particles with wall velocity, %zu boundary cells",
               pv->local()->size(), pv->getCName(), bc.size());

        const int nthreads = 64;
        SAFE_KERNEL_LAUNCH(
                bounce_kernels::sdfBounce,
                getNblocks(bc.size(), nthreads), nthreads, 0, stream,
                view, cl->cellInfo(), bc.devPtr(), bc.size(), dt,
                this->insideWallChecker_.handler(),
                velField_.handler(),
                this->bounceForce_.devPtr());

        CUDA_Check( hipPeekAtLastError() );
    }
}


template class WallWithVelocity<StationaryWallSphere,   VelocityFieldRotate>;
template class WallWithVelocity<StationaryWallCylinder, VelocityFieldRotate>;
template class WallWithVelocity<StationaryWallPlane,    VelocityFieldTranslate>;
template class WallWithVelocity<StationaryWallPlane,    VelocityFieldOscillate>;

} // namespace mirheo
